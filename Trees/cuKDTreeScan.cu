#include "hip/hip_runtime.h"

#ifdef _DEBUG
#define NUTTY_DEBUG
#endif

#undef NUTTY_DEBUG

#include <cutil_math.h>
#include "cuKDTree.h"
#include "kd_kernel.h"
#include "kd_scan_kernel.h"
#include "shared_kernel.h"
#include "shared_types.h"
#include <Reduce.h>
#include <Sort.h>
#include <Scan.h>
#include <queue>
#include <ForEach.h>
#include <Fill.h>
#include <Functions.h>
#include <cuda/Globals.cuh>
#include "buffer_print.h"
#include <chimera/Timer.h>
#include <fstream>

#define NODES_GROUP_SIZE 256U
#define EVENT_GROUP_SIZE 256U

struct cudaErrorBuffer
{
    CTuint* devMemory;

    cudaErrorBuffer(void)
    {
        hipMalloc(&devMemory, 4 * sizeof(CTuint));

        CTuint null = 0;
        hipMemcpy(devMemory, &null, 4, hipMemcpyHostToDevice);
    }

    bool check(void)
    {
        CTuint hostMemory[4];
        hipMemcpy(&hostMemory, devMemory, 4 * sizeof(CTuint), hipMemcpyDeviceToHost);

        if(hostMemory[0])
        {
            __ct_printf("GOT ERROR = %d %d %d %d\n", hostMemory[0], hostMemory[1], hostMemory[2], hostMemory[3]);
            //__debugbreak();
            return true;
        }

        CTuint null = 0;
        hipMemcpy(devMemory, &null, 4, hipMemcpyHostToDevice);

        return false;
    }

    ~cudaErrorBuffer(void)
    {
        hipFree(devMemory);
    }
};

template<>
struct ShrdMemory<CTuint3>
{
    __device__ CTuint3* Ptr(void) 
    { 
        extern __device__ __shared__ CTuint3 s_b4[];
        return s_b4;
    }
};

#undef PROFILE
#ifdef PROFILE
#define PROFILE_START chimera::util::HTimer timer; hipDeviceSynchronize(); timer.Start()
#define PROFILE_END hipDeviceSynchronize(); timer.Stop(); g_time += timer.GetMillis()
#else
#define PROFILE_START
#define PROFILE_END
#endif

#undef PRINT_OUT
#ifndef _DEBUG
#undef PRINT_OUT
#endif

#ifndef PRINT_OUT
#undef PRINT_BUFFER(_name)
#undef PRINT_BUFFER_N(_name, _tmp)
#undef PRINT_RAW_BUFFER
#undef PRINT_RAW_BUFFER_N
#undef ct_printf

#define PRINT_BUFFER(_name)
#define PRINT_BUFFER_N(_name, _tmp)
#define PRINT_RAW_BUFFER(_name)
#define PRINT_RAW_BUFFER_N(_name, _N)
#define ct_printf(...)
#endif

template<>
struct ShrdMemory<IndexedSAHSplit>
{
    __device__ IndexedSAHSplit* Ptr(void) 
    { 
        extern __device__ __shared__ IndexedSAHSplit s_split[];
        return s_split;
    }
};

template<>
struct ShrdMemory<BBox>
{
    __device__ BBox* Ptr(void) 
    { 
        extern __device__ __shared__ BBox s_bbox[];
        return s_bbox;
    }
};

template<>
struct ShrdMemory<IndexedEvent>
{
    __device__ IndexedEvent* Ptr(void) 
    { 
        extern __device__ __shared__ IndexedEvent s_edge[];
        return s_edge;
    }
};

struct AxisSort
{
    char axis;
    AxisSort(char a) : axis(a)
    {

    }
    __device__ __host__ char operator()(const CTreal3& f0, const CTreal3& f1)
    {
        return getAxis(f0, axis) > getAxis(f1, axis);
    }
};

struct float3min
{
    __device__ float3 operator()(const float3& t0, const float3& t1)
    {
        float3 r;
        r.x = nutty::binary::Min<float>()(t0.x, t1.x);
        r.y = nutty::binary::Min<float>()(t0.y, t1.y);
        r.z = nutty::binary::Min<float>()(t0.z, t1.z);
        return r;
    }
};

struct float3max
{
    __device__  float3 operator()(const float3& t0, const float3& t1)
    {
        float3 r;
        r.x = nutty::binary::Max<float>()(t0.x, t1.x);
        r.y = nutty::binary::Max<float>()(t0.y, t1.y);
        r.z = nutty::binary::Max<float>()(t0.z, t1.z);
        return r;
    }
};

struct ReduceBBox
{
    __device__  BBox operator()(const BBox& t0, const BBox& t1)
    {
        BBox bbox;
        bbox.m_min = fminf(t0.m_min, t1.m_min);
        bbox.m_max = fmaxf(t0.m_max, t1.m_max);
        return bbox;
    }
};

template <
    typename T
>
struct InvTypeOp
{
    __device__ T operator()(T elem)
    {
        return (elem < 2) * (elem ^ 1);
    }

    T GetNeutral(void)
    {
        return 0;
    }
};

template <
    typename T
>
struct TypeOp
{
    __device__ CTuint operator()(T elem)
    {
        return (elem < 2) * elem;
    }

    __device__ __host__ CTuint GetNeutral(void)
    {
        return 0;
    }
};

template <
    typename T
>
struct EventStartScanOp
{
    __device__ T operator()(T elem)
    {
        return elem ^ 1;
    }

    __device__ __host__ T GetNeutral(void)
    {
        return 1;
    }
};

struct ScanByte3
{
    __device__ CTuint3 operator()(CTbyte3 elem)
    {
        CTuint3 v;
        v.x = elem.x ^ 1;
        v.y = elem.y ^ 1;
        v.z = elem.z ^ 1;
        return v;
    }

    __device__ __host__ CTbyte3 GetNeutral(void)
    {
        CTbyte3 v;
        v.x = 1; v.y = 1; v.z = 1;
        return v;
    }
};

template <
    typename T
>
struct EventEndScanOp
{
    __device__ T operator()(T elem)
    {
        return elem;
    }

    T GetNeutral(void)
    {
        return 0;
    }
};

double g_time = 0;

void PrintEventLine(EventLine& line, CTuint l)
{
    ct_printf("PrintEventLine\n");
//    PRINT_BUFFER_N(line.indexedEvent[line.toggleIndex], l);
    ///PRINT_BUFFER_N(line.nodeIndex[line.toggleIndex], l);
//     PRINT_BUFFER_N(line.prefixSum[line.toggleIndex], l);
//     PRINT_BUFFER_N(line.primId[line.toggleIndex], l);
//     PRINT_BUFFER_N(line.type[line.toggleIndex], l);
    ct_printf("End\n");
}

template <typename Operator, typename T>
void ScanBinaryTriples(ConstTuple<3, T>& src, Tuple<3, CTuint>& scanned, Tuple<3, CTuint>& sums, CTuint N, Operator op, hipStream_t pStream)
{
    static const CTuint block = 256;

    ConstTuple<3, CTuint> constSums;
    constSums.ts[0] = sums.ts[0];
    constSums.ts[1] = sums.ts[1];
    constSums.ts[2] = sums.ts[2];

    CTuint grid = nutty::cuda::GetCudaGrid(N, block);

    binaryTripleGroupScan<block><<<grid, block, 0, pStream>>>(
        src, scanned, sums, op,
        N);
    
    DEVICE_SYNC_CHECK();

    CTuint sumsCount = nutty::cuda::GetCudaGrid(N, block);

    if(sumsCount > 1)
    {
#if 1
        nutty::PrefixSumOp<CTuint> _op;
        completeScan2<256, 3><<<3, 256, 0, pStream>>>(constSums, sums, _op, sumsCount);

        DEVICE_SYNC_CHECK();
#else
        CTuint shrdStepElemperThread = nutty::cuda::GetCudaGrid(sumsCount, 256U);

        switch(shrdStepElemperThread)
        {
        case  1: completeScan<256, 3, 1><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  2: completeScan<256, 3, 2><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  3: completeScan<256, 3, 3><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  4: completeScan<256, 3, 4><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  5: completeScan<256, 3, 5><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  6: completeScan<256, 3, 6><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  7: completeScan<256, 3, 7><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  8: completeScan<256, 3, 8><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case  9: completeScan<256, 3, 9><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case 10: completeScan<256, 3, 10><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case 11: completeScan<256, 3, 11><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        case 12: completeScan<256, 3, 12><<<3, 256, 0, pStream>>>(constSums, sums, op, sumsCount); break;
        default:   __ct_printf("error\n"); exit(0); break;
        };
#endif

         spreadScannedSums<<<grid-1, block, 0, pStream>>>(scanned, sums, N);
         DEVICE_SYNC_CHECK();
    }
}

void cuKDTreeScan::InitBuffer(void)
{
    CTuint primitiveCount = m_orginalVertices.Size() / 3;

    m_depth = (byte)min(64, max(1, (m_depth == 0xFF ? GenerateDepth(primitiveCount) : m_depth)));

    m_primAABBs.Resize(primitiveCount); nutty::ZeroMem(m_primAABBs);

    for(int i = 0; i < 3; ++i)
    {
       // m_events3[i].SetNodeIndexBuffer(&m_eventNodeIndex);
        m_eventLines.eventLines[i].SetNodeIndexBuffer(&m_eventNodeIndex);
    }

    GrowNodeMemory();
    GrowPerLevelNodeMemory(64);
    GrowSplitMemory(4 * primitiveCount);

    ClearBuffer();

    m_dthAsyncIntCopy.Init(2);
    m_dthAsyncByteCopy.Init(2);

    m_dthAsyncNodesContent.Init(100);

    m_gotLeaves.Resize(1);
}

void cuKDTreeScan::ClearBuffer(void)
{
    nutty::ZeroMem(m_nodesBBox[0]);
    nutty::ZeroMem(m_nodesBBox[1]);

    nutty::ZeroMem(m_nodes_ContentCount);
    nutty::ZeroMem(m_nodes_IsLeaf);
    nutty::ZeroMem(m_nodes_Split);
    nutty::ZeroMem(m_nodes_ContentStartAdd);
    nutty::ZeroMem(m_nodes_SplitAxis);
    nutty::ZeroMem(m_nodes_LeftChild);
    nutty::ZeroMem(m_nodes_RightChild);

    nutty::ZeroMem(m_splits_Above);
    nutty::ZeroMem(m_splits_Below);
    nutty::ZeroMem(m_splits_Axis);
    nutty::ZeroMem(m_splits_Plane);

    nutty::ZeroMem(m_leafNodesContentCount);
    nutty::ZeroMem(m_leafNodesContentStart);
}

void cuKDTreeScan::GrowPerLevelNodeMemory(CTuint newSize)
{
    m_activeNodesIsLeaf.Resize(newSize);
    m_activeNodes.Resize(newSize);
    m_activeNodesThisLevel.Resize(newSize);
    m_newActiveNodes.Resize(newSize);
    m_nodesBBox.Resize(newSize);
    m_nodes_ContentStartAdd.Resize(newSize);
    m_nodes_ContentCount.Resize(newSize);

    m_nodes.isLeaf = m_nodes_IsLeaf.GetDevicePtr()();
    m_nodes.splitAxis = m_nodes_SplitAxis.GetDevicePtr()();
    m_nodes.split = m_nodes_Split.GetDevicePtr()();
    m_nodes.contentStart = m_nodes_ContentStartAdd.GetDevicePtr()();
    m_nodes.contentCount = m_nodes_ContentCount.GetDevicePtr()();
    m_nodes.leftChild = m_nodes_LeftChild.GetDevicePtr()();
    m_nodes.rightChild = m_nodes_RightChild.GetDevicePtr()();
    m_nodes.nodeToLeafIndex = m_nodes_NodeIdToLeafIndex.GetDevicePtr()();

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbolAsync(HIP_SYMBOL(g_nodes), &m_nodes, sizeof(Node), 0, hipMemcpyHostToDevice, m_pStream));
}

void cuKDTreeScan::GrowNodeMemory(void)
{
    size_t newSize = m_nodes_IsLeaf.Size() ? m_nodes_IsLeaf.Size() * 4 : 32;

    m_nodes_IsLeaf.Resize(newSize);
    m_nodes_Split.Resize(newSize);
    m_nodes_NodeIdToLeafIndex.Resize(newSize);
    m_nodes_SplitAxis.Resize(newSize);
    m_nodes_LeftChild.Resize(newSize);
    m_nodes_RightChild.Resize(newSize);

    m_nodes.isLeaf = m_nodes_IsLeaf.GetDevicePtr()();
    m_nodes.splitAxis = m_nodes_SplitAxis.GetDevicePtr()();
    m_nodes.split = m_nodes_Split.GetDevicePtr()();
    m_nodes.contentStart = m_nodes_ContentStartAdd.GetDevicePtr()();
    m_nodes.contentCount = m_nodes_ContentCount.GetDevicePtr()();
    m_nodes.leftChild = m_nodes_LeftChild.GetDevicePtr()();
    m_nodes.rightChild = m_nodes_RightChild.GetDevicePtr()();
    m_nodes.nodeToLeafIndex = m_nodes_NodeIdToLeafIndex.GetDevicePtr()();

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbolAsync(HIP_SYMBOL(g_nodes), &m_nodes, sizeof(Node), 0, hipMemcpyHostToDevice, m_pStream));
}

void cuKDTreeScan::GrowSplitMemory(CTuint eventCount)
{
    m_splits_Above.Resize(eventCount);
    m_splits_Below.Resize(eventCount);
    m_splits_Axis.Resize(eventCount);
    m_splits_Plane.Resize(eventCount);
    m_splits_IndexedSplit.Resize(eventCount);

    m_eventIsLeaf.Resize(eventCount);
    
    m_splits.above = m_splits_Above.GetDevicePtr()();
    m_splits.below = m_splits_Below.GetDevicePtr()();
    m_splits.axis = m_splits_Axis.GetDevicePtr()();
    m_splits.indexedSplit = m_splits_IndexedSplit.GetDevicePtr()();
    m_splits.v = m_splits_Plane.GetDevicePtr()();

    m_eventNodeIndex.Resize(eventCount);

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbolAsync(HIP_SYMBOL(g_splits), &m_splits, sizeof(Split), 0, hipMemcpyHostToDevice, m_pStream));
    
    SplitConst splitsConst;
    splitsConst.above = m_splits_Above.GetDevicePtr()();
    splitsConst.below = m_splits_Below.GetDevicePtr()();
    splitsConst.axis = m_splits_Axis.GetDevicePtr()();
    splitsConst.indexedSplit = m_splits_IndexedSplit.GetDevicePtr()();
    splitsConst.v = m_splits_Plane.GetDevicePtr()();

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbolAsync(HIP_SYMBOL(g_splitsConst), &splitsConst, sizeof(SplitConst), 0, hipMemcpyHostToDevice, m_pStream));
}

void cuKDTreeScan::PrintStatus(const char* msg /* = NULL */)
{
    ct_printf("PrintStatus: %s\n", msg == NULL ? "" : msg);
    PRINT_BUFFER(m_nodes_ContentCount);
    PRINT_BUFFER(m_nodes_ContentStartAdd);
}

void cuKDTreeScan::ScanEventTypesTriples(CTuint eventCount)
{
    CTbyte add = m_eventLines.toggleIndex;

    ConstTuple<3, CTbyte> ptr;
    ptr.ts[0] = m_eventLines.eventLines[0].type[add].GetConstPointer();
    ptr.ts[1] = m_eventLines.eventLines[1].type[add].GetConstPointer();
    ptr.ts[2] = m_eventLines.eventLines[2].type[add].GetConstPointer();

    Tuple<3, CTuint> ptr1;
    ptr1.ts[0] = m_eventLines.eventLines[0].typeStartScanned.GetPointer();
    ptr1.ts[1] = m_eventLines.eventLines[1].typeStartScanned.GetPointer();
    ptr1.ts[2] = m_eventLines.eventLines[2].typeStartScanned.GetPointer();
    
    Tuple<3, CTuint> sums;
    sums.ts[0] = m_eventLines.eventLines[0].scannedEventTypeEndMaskSums.GetPointer();
    sums.ts[1] = m_eventLines.eventLines[1].scannedEventTypeEndMaskSums.GetPointer();
    sums.ts[2] = m_eventLines.eventLines[2].scannedEventTypeEndMaskSums.GetPointer();

    nutty::PrefixSumOp<CTbyte> op;
    ScanBinaryTriples(ptr, ptr1, sums, eventCount, op, m_pStream); 
}

void cuKDTreeScan::ComputeSAH_Splits(
    CTuint nodeCount,
    CTuint eventCount, 
    const CTuint* nodesContentCount)
{
    CTuint eventBlock = EVENT_GROUP_SIZE;
    CTuint eventGrid = nutty::cuda::GetCudaGrid(eventCount, eventBlock);
    //cuEventLineTriple tripleLine(m_events3, 0);
    CTuint start = 0;

    //m_pool.Reset();
    m_typeScanner.Resize(eventCount);
    m_types3.Resize(eventCount);
    
#if 0
     for(CTbyte i = 0; i < 3; ++i)
     {
//        // m_eventLines.eventLines[i].Resize(eventCount);
         m_eventLines.eventLines[i].ScanEventTypes(eventCount);
//         //PRINT_RAW_BUFFER_N(m_eventLines.eventLines[i].typeStartScanner.GetPrefixSum(), eventCount);
// //         PRINT_RAW_BUFFER(m_events3[i].tmpType);
// //         OutputDebugStringA("\n");
//         //nutty::ZeroMem(m_eventLines.eventLines[i].typeStartScanned);
//        
     }
#endif

    
//     static EventStartScanOp<CTbyte> op0;
//     for(CTbyte k = 0; k < 3; ++k)
//     {
//         groupScan<256U, CTbyte, CTuint, EventStartScanOp<CTbyte>> <<<eventGrid, eventBlock>>>(
//             m_eventLines.eventLines[k].type[m_eventLines.toggleIndex].GetConstPointer(),
//             m_eventLines.eventLines[k].typeStartScanned.GetPointer(), 
//             sums.GetPointer(), 
//             op0, eventCount);
//     }

     //nutty::ZeroMem(m_eventLines.eventLines[0].scannedEventTypeEndMaskSums);

     ScanEventTypesTriples(eventCount);
     DEVICE_SYNC_CHECK();
#if 0
    for(CTbyte i = 0; i < 3; ++i)
    {
        nutty::HostBuffer<CTuint> tmp0(eventCount);
        nutty::HostBuffer<CTuint> tmp1(eventCount);
        nutty::Copy(tmp0.Begin(), m_eventLines.eventLines[i].typeStartScanner.GetPrefixSum().Begin(), m_eventLines.eventLines[i].typeStartScanner.GetPrefixSum().Begin() + eventCount);
        nutty::Copy(tmp1.Begin(), m_eventLines.eventLines[i].typeStartScanned.Begin(), m_eventLines.eventLines[i].typeStartScanned.Begin() + eventCount);
        for(int k = 0; k < eventCount; ++k)
        {
            if(tmp1[k] != tmp0[k])
            {
                __ct_printf("error: %d %d %d %d\n", tmp1[k], tmp0[k], k, i);
                //exit(0);
                    const CTuint block = 512; //nutty::cuda::GetCudaBlock(N, 256U);
                CTuint grid = nutty::cuda::GetCudaGrid(eventCount, block);
                size_t sumSize = (eventCount % nutty::cuda::SCAN_ELEMS_PER_BLOCK) == 0 ? eventCount / nutty::cuda::SCAN_ELEMS_PER_BLOCK : (eventCount / nutty::cuda::SCAN_ELEMS_PER_BLOCK) + 1;
                PRINT_RAW_BUFFER_N(m_eventLines.eventLines[i].scannedEventTypeEndMaskSums, sumSize);
                PRINT_RAW_BUFFER_N(m_eventLines.eventLines[i].typeStartScanner.m_scannedSums, sumSize);
                exit(0);
            }
        }
    }
    #endif

    DEVICE_SYNC_CHECK();
    const CTuint elemsPerThread = 1;
    CTuint N = eventCount;//nutty::cuda::GetCudaGrid(eventCount, elemsPerThread);
    CTuint sahBlock = EVENT_GROUP_SIZE;
    CTuint sahGrid = nutty::cuda::GetCudaGrid(N, sahBlock);

    computeSAHSplits3<1, elemsPerThread><<<sahGrid, sahBlock, 0, m_pStream>>>(
        nodesContentCount,
        m_nodes_ContentStartAdd.Begin()(),
        m_nodesBBox[0].Begin()(),
        eventCount,
        m_eventLines.toggleIndex);

//     computeSAHSplits3Old<<<sahGrid, sahBlock, 0, m_pStream>>>(
//         nodesContentCount,
//         m_nodes_ContentStartAdd.Begin()(),
//         m_nodesBBox[0].Begin()(),
//         eventCount,
//         m_eventLines.toggleIndex);

    DEVICE_SYNC_CHECK();

#if 0
    for(int i = 0; i < eventCount; ++i)
    {
            ct_printf("%d [%d %d] id=%d Axis=%d, Plane=%f SAH=%f :: \n", 
                i, m_splits_Below[i], m_splits_Above[i],
                m_splits_IndexedSplit[i].index, 
                (CTuint)m_splits_Axis[i],
                m_splits_Plane[i], 
                (m_splits_IndexedSplit[i].sah == INVALID_SAH ? -1 : m_splits_IndexedSplit[i].sah));

            //BBox bbox = m_nodesBBox[0][ m_events3[0].nodeIndex[m_events3[0].toggleIndex][i] ];
            //ct_printf("%f %f %f | %f %f %f\n", bbox.m_min.x, bbox.m_min.y, bbox.m_min.z, bbox.m_max.x, bbox.m_max.y, bbox.m_max.z);
    }
#endif

    //start = 0;
    //m_pool.Reset();

    if(true) //5
    {
        if(nodeCount == 1)
        {
            IndexedSAHSplit neutralSplit;
            neutralSplit.index = 0;
            neutralSplit.sah = FLT_MAX;
            nutty::Reduce(m_splits_IndexedSplit.Begin(), m_splits_IndexedSplit.Begin() + eventCount, ReduceIndexedSplit(), neutralSplit, m_pStream);
            DEVICE_SYNC_CHECK();
        }
#if defined DYNAMIC_PARALLELISM
//         else if(nodeCount == 2)
//         {
//             CTuint nodeBlock = nodeCount;//nutty::cuda::GetCudaBlock(nodeCount, 32U);
//             CTuint nodeGrid = 1;
//             dpReduceSAHSplits<<<nodeGrid, nodeBlock>>>(m_splits_IndexedSplit.GetPointer(), nodeCount);
//         }
#endif
        else
        {
            const CTuint blockSize = 512U;
            CTuint N = nodeCount * blockSize;
            CTuint reduceGrid = nutty::cuda::GetCudaGrid(N, blockSize);
            //cudaErrorBuffer errorBuffer;

            segReduce<blockSize><<<reduceGrid, blockSize, 0, m_pStream>>>(m_splits_IndexedSplit.GetPointer(), N, eventCount);

//             if(errorBuffer.check())
//             {
//                 PrintBuffer(m_nodes_ContentCount, nodeCount);
//                 PrintBuffer(m_nodes_ContentStartAdd, nodeCount);
//                 __debugbreak();
//             }
            DEVICE_SYNC_CHECK();
        }

#if 0
        m_hNodesContentCount.Resize(nodeCount);
        nutty::Copy(m_hNodesContentCount.Begin(), m_nodes_ContentCount.Begin(), nodeCount);
        for(int i = 0; i < nodeCount; ++i)
        {
            CTuint cc = m_hNodesContentCount[i];
            CTuint length = 2 * cc;

            IndexedSAHSplit s = *(m_splits_IndexedSplit.Begin() + start);

            if(IS_INVALD_SAH(s.sah))
            {
                std::stringstream ss;
                ss << m_nodesBBox[0][i];
                __ct_printf("%s ", ss.str().c_str());
                __ct_printf("id=%d, memoryadd=%d ", s.index, start);
                CTreal plane = m_splits_Plane[s.index];
                CTbyte axis = m_splits_Axis[s.index];
                CTuint below = m_splits_Below[s.index];
                CTuint above = m_splits_Above[s.index];
                __ct_printf("contentCount=%d axis=%d plane=%f sah=%f below=%d above=%d\n", cc, (CTuint)axis, plane, s.sah, below, above);

                for(int a = start; a < start + length; ++a)
                {
                    std::stringstream ss;
                    ss << m_nodesBBox[0][i];
                        __ct_printf("%d [%d %d] id=%d Axis=%d, Plane=%f SAH=%f :: %s\n", 
                            a, m_splits_Below[a], m_splits_Above[a],
                            m_splits_IndexedSplit[a].index, 
                            (CTuint)m_splits_Axis[a],
                            m_splits_Plane[a], 
                            (m_splits_IndexedSplit[a].sah == INVALID_SAH ? -1 : m_splits_IndexedSplit[a].sah), ss.str().c_str());
                }

            }

            start += length;
        }
#endif
    }
    else
    {
        //m_dthAsyncNodesContent.WaitForCopy();
        m_hNodesContentCount.Resize(nodeCount);
        nutty::Copy(m_hNodesContentCount.Begin(), m_nodes_ContentCount.Begin(), nodeCount);

        for(int i = 0; i < nodeCount; ++i)
        {
            CTuint cc = m_hNodesContentCount[i];
            CTuint length = 2 * cc;
    #ifdef _DEBUG
            if(cc <= MAX_ELEMENTS_PER_LEAF)
            {
                assert(0 && "cc <= MAX_ELEMENTS_PER_LEAF");
                //start += length;
                continue;
            }
    #endif
            IndexedSAHSplit neutralSplit;
            neutralSplit.index = 0;
            neutralSplit.sah = FLT_MAX;
        
//             nutty::cuStream& stream = m_pool.PeekNextStream();
//             nutty::SetStream(stream);

            nutty::Reduce(m_splits_IndexedSplit.Begin() + start, m_splits_IndexedSplit.Begin() + start + length, ReduceIndexedSplit(), neutralSplit, m_pStream);

            DEVICE_SYNC_CHECK();
    #ifdef PRINT_OUT
            IndexedSAHSplit s = *(m_splits_IndexedSplit.Begin() + start);
            std::stringstream ss;
            ss << m_nodesBBox[0][i];
            ct_printf("%s ", ss.str().c_str());
            ct_printf("id=%d, memoryadd=%d ", s.index, start);
            CTreal plane = m_splits_Plane[s.index];
            CTbyte axis = m_splits_Axis[s.index];
            CTuint below = m_splits_Below[s.index];
            CTuint above = m_splits_Above[s.index];
            ct_printf("axis=%d plane=%f sah=%f below=%d above=%d\n", (CTuint)axis, plane, s.sah, below, above);

            if(IS_INVALD_SAH(s.sah))
            {
                for(int i = start; i < start + length; ++i)
                {
                        ct_printf("%d [%d %d] id=%d Axis=%d, Plane=%f SAH=%f :: ", 
                            i, m_splits_Below[i], m_splits_Above[i],
                            m_splits_IndexedSplit[i].index, 
                            (CTuint)m_splits_Axis[i],
                            m_splits_Plane[i], 
                            (m_splits_IndexedSplit[i].sah == INVALID_SAH ? -1 : m_splits_IndexedSplit[i].sah));

                        BBox bbox;// = m_nodesBBox[0][ m_events3[0].nodeIndex[m_events3[0].toggleIndex][i] ];
                        ct_printf("%f %f %f | %f %f %f\n", bbox.m_min.x, bbox.m_min.y, bbox.m_min.z, bbox.m_max.x, bbox.m_max.y, bbox.m_max.z);
                }
                __debugbreak();
            }
    #endif
            start += length;
        }

//         for(CTuint i = 0; i < min(m_pool.GetStreamCount(), nodeCount); ++i)
//         {
//             nutty::cuStream& stream = m_pool.GetStream(i);
//             nutty::cuEvent e = stream.RecordEvent();
//             hipStreamWaitEvent(0, e.GetPointer(), 0);
//         }
//     
//         nutty::SetDefaultStream();
    }
}

CTuint cuKDTreeScan::CheckRangeForLeavesAndPrepareBuffer(nutty::DeviceBuffer<CTbyte>::iterator& isLeafBegin, CTuint nodeOffset, CTuint nodeRange)
{
    m_leafCountScanner.Resize(nodeRange);

    m_leafCountScanner.ExcScan(isLeafBegin + nodeOffset, isLeafBegin + nodeOffset + nodeRange, TypeOp<CTbyte>(), m_pStream);

    DEVICE_SYNC_CHECK();

    m_dthAsyncIntCopy.WaitForStream(m_stream);
    m_dthAsyncByteCopy.WaitForStream(m_stream);

    m_dthAsyncIntCopy.StartCopy(m_leafCountScanner.GetPrefixSum().GetConstPointer() + nodeRange - 1, 0);
    m_dthAsyncByteCopy.StartCopy(isLeafBegin() + nodeOffset + nodeRange - 1, 0);

    CTuint block = NODES_GROUP_SIZE; //nutty::cuda::GetCudaBlock(nodeRange, 256U);
    CTuint grid = nutty::cuda::GetCudaGrid(nodeRange, block);

    DEVICE_SYNC_CHECK();

    if(m_interiorCountScanned.Size() <= nodeRange)
    {
        m_interiorCountScanned.Resize(nodeRange);
        m_maskedInteriorContent.Resize(nodeRange);
        m_interiorContentScanner.Resize(nodeRange);
        m_leafContentScanned.Resize(nodeRange);
    }

    createInteriorContentCountMasks<<<grid, block, 0, m_pStream>>>(
        isLeafBegin() + nodeOffset,
        m_nodes_ContentCount.Begin()(), 
        m_maskedInteriorContent.Begin()(), nodeRange);

    DEVICE_SYNC_CHECK();

    m_interiorContentScanner.ExcScan(m_maskedInteriorContent.Begin(), m_maskedInteriorContent.Begin() + nodeRange, nutty::PrefixSumOp<CTuint>(), m_pStream);
    
    DEVICE_SYNC_CHECK();

    makeOthers<<<grid, block, 0, m_pStream>>>(

        m_nodes_ContentStartAdd.Begin()(), 
        m_interiorContentScanner.GetPrefixSum().Begin()(), 
        m_leafContentScanned.Begin()(), 
        
        m_leafCountScanner.GetPrefixSum().Begin()(), 
        m_interiorCountScanned.Begin()(), 
        
        nodeRange);

    DEVICE_SYNC_CHECK();

    m_dthAsyncIntCopy.WaitForCopy();
    m_dthAsyncByteCopy.WaitForCopy();
    CTuint leafCount = m_dthAsyncIntCopy[0] + (m_dthAsyncByteCopy[0] == 1);

    DEVICE_SYNC_CHECK();

    return leafCount;
}

MakeLeavesResult cuKDTreeScan::MakeLeaves(
    nutty::DeviceBuffer<CTbyte>::iterator& isLeafBegin,
    CTuint g_nodeOffset, 
    CTuint nodeOffset, 
    CTuint nodeCount, 
    CTuint eventCount, 
    CTuint currentLeafCount, 
    CTuint leafContentOffset,
    CTuint initNodeToLeafIndex,
    CTbyte gotLeaves)
{

    CTuint leafCount = 0;
    if(gotLeaves)
    {
         leafCount = CheckRangeForLeavesAndPrepareBuffer(isLeafBegin, nodeOffset, nodeCount);

         DEVICE_SYNC_CHECK();
    }
     
    if(!leafCount)
    {
        MakeLeavesResult result;
        result.leafCount = 0;
        result.interiorPrimitiveCount = eventCount/2;
        result.leafPrimitiveCount = 0;
        return result;
    }
    
    m_dthAsyncIntCopy.WaitForStream(m_stream);
    m_dthAsyncByteCopy.WaitForStream(m_stream);

    m_dthAsyncIntCopy.StartCopy(m_leafContentScanned.GetConstPointer() + nodeCount - 1, 0);
    m_dthAsyncIntCopy.StartCopy(m_nodes_ContentCount.GetConstPointer() + nodeCount - 1, 1);
    m_dthAsyncByteCopy.StartCopy(m_activeNodesIsLeaf.GetConstPointer() + nodeCount + nodeOffset - 1, 0);
 
    m_leafNodesContentStart.Resize(currentLeafCount + leafCount);
    m_leafNodesContentCount.Resize(currentLeafCount + leafCount);

    const CTuint eventBlock = EVENT_GROUP_SIZE;
    CTuint eventGrid = nutty::cuda::GetCudaGrid(eventCount, eventBlock);

    CTuint nodeBlock = NODES_GROUP_SIZE;
    CTuint nodeGrid = nutty::cuda::GetCudaGrid(nodeCount, nodeBlock);
    

#if 1
//     m_eventIsLeafScanner.Resize(eventCount);
//     m_eventIsLeafScanner.ExcScan(m_eventIsLeaf.Begin(), m_eventIsLeaf.Begin() + eventCount, TypeOp<CTbyte>());

    m_eventIsLeafScanned.Resize(eventCount);
    m_eventIsLeafScannedSums.Resize(eventCount/256 + 256);

    binaryGroupScan<256><<<eventGrid, eventBlock, 0, m_pStream>>>(
        m_eventIsLeaf.GetConstPointer(), m_eventIsLeafScanned.GetPointer(), m_eventIsLeafScannedSums.GetPointer(), TypeOp<CTbyte>(), eventCount);

    DEVICE_SYNC_CHECK();

    CTuint sumsCount = nutty::cuda::GetCudaGrid(eventCount, EVENT_GROUP_SIZE);

    if(sumsCount > 1)
    {
        nutty::PrefixSumOp<CTuint> _op;
        completeScan<256><<<1, 256, 0, m_pStream>>>(m_eventIsLeafScannedSums.GetConstPointer(), m_eventIsLeafScannedSums.GetPointer(), _op, sumsCount);

        DEVICE_SYNC_CHECK();

        spreadScannedSumsSingle<<<eventGrid-1, eventBlock, 0, m_pStream>>>(
                m_eventIsLeafScanned.GetPointer(), m_eventIsLeafScannedSums.GetConstPointer(), eventCount);
    }

#endif
    
    DEVICE_SYNC_CHECK();

    if(m_leafNodesContent.Size() < leafContentOffset + eventCount/2)
    {
        m_leafNodesContent.Resize(leafContentOffset + eventCount/2);
    }

    DEVICE_SYNC_CHECK();

    compactMakeLeavesData<<<eventGrid, eventBlock, 0, m_pStream>>>(
        isLeafBegin() + nodeOffset,
        m_interiorCountScanned.GetPointer(),
        m_leafContentScanned.GetPointer(),
         
        m_eventIsLeafScanned.GetConstPointer(),
        //m_eventIsLeafScanner.GetPrefixSum().GetConstPointer(),
            
        m_nodes_ContentCount.GetPointer(),
        m_eventIsLeaf.GetPointer(),

        m_leafCountScanner.GetPrefixSum().GetConstPointer(), 
        m_interiorCountScanned.GetConstPointer(),

        m_activeNodes.GetPointer(),
        m_leafCountScanner.GetPrefixSum().GetConstPointer(),
        m_interiorContentScanner.GetPrefixSum().GetConstPointer(),
        m_nodesBBox[1].GetPointer(),

        m_leafNodesContent.GetPointer(),
        m_nodes_NodeIdToLeafIndex.GetPointer(),
        m_newNodesContentCount.GetPointer(),
        m_newNodesContentStartAdd.GetPointer(),
        m_leafNodesContentStart.GetPointer(),
        m_leafNodesContentCount.GetPointer(),
        m_newActiveNodes.GetPointer(),
        m_nodesBBox[0].GetPointer(),
         
        g_nodeOffset,
        leafContentOffset,
        currentLeafCount,
        nodeCount,
        m_eventLines.toggleIndex,
        eventCount);

    DEVICE_SYNC_CHECK();

    m_eventLines.Toggle();

    CTuint copyDistance = nodeCount - leafCount;

    if(copyDistance)
    {
        CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_nodes_ContentCount.GetPointer(), m_newNodesContentCount.GetPointer(), nodeCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));
        CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_nodes_ContentStartAdd.GetPointer(), m_newNodesContentStartAdd.GetPointer(), nodeCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));
        CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_activeNodes.GetPointer(), m_newActiveNodes.GetPointer(), nodeCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));
    }

    m_dthAsyncIntCopy.WaitForCopy();
    m_dthAsyncByteCopy.WaitForCopy();

    CTuint leafPrimCount = m_dthAsyncIntCopy[0] + m_dthAsyncByteCopy[0] * m_dthAsyncIntCopy[1];

    CTuint interiorPrimCount = eventCount/2 - leafPrimCount;
    interiorPrimCount = interiorPrimCount > eventCount/2 ? 0 : interiorPrimCount;

    MakeLeavesResult result;
    result.leafCount = leafCount;
    result.interiorPrimitiveCount = interiorPrimCount;
    result.leafPrimitiveCount = leafPrimCount;

    DEVICE_SYNC_CHECK();

    return result;
}

void ClipMask::Resize(size_t size, hipStream_t pStream)
{
    if(mask[0].Size() >= size) return;
    size = (CTuint)(1.2 * size);
    //mask3.Resize(size);
    mask3Scanner.Resize(size);
    for(int i = 0; i < 3; ++i)
    {
        scannedMask[i].Resize(size);
        scannedSums[i].Resize(size);
        mask[i].Resize(size); 
        newSplits[i].Resize(size);
        index[i].Resize(size);
//        maskScanner[i].Resize(size);
    }
    cuClipMaskArray mm;
    GetPtr(mm);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(g_clipArray), &mm, sizeof(cuClipMaskArray), 0, hipMemcpyHostToDevice, pStream);

    cuConstClipMask cmss[3];
    GetConstPtr(cmss[0], 0);
    GetConstPtr(cmss[1], 1);
    GetConstPtr(cmss[2], 2);

    hipMemcpyToSymbolAsync(HIP_SYMBOL(cms), &cmss, 3 * sizeof(cuConstClipMask), 0, hipMemcpyHostToDevice, pStream);
}

void EventLine::ScanEvents(CTuint length)
{
    __ct_printf("fatal error: ScanEvents not working\n");
    exit(-1);
    //eventScanner.ExcScan(mask.Begin(), mask.Begin() + length, nutty::PrefixSumOp<CTbyte>());
}

struct ClipMaskPrefixSumOP
{
    __device__ CTuint operator()(CTbyte elem)
    {
        return isSet(elem) ? 1 : 0;
    }

    __device__ __host__ CTuint GetNeutral(void)
    {
        return 0;
    }
};

struct ClipMaskPrefixSum3OP
{
    __device__ CTuint3 operator()(CTbyte3 elem)
    {
        CTuint3 v;
        v.x = isSet(elem.x) ? 1 : 0;
        v.y = isSet(elem.y) ? 1 : 0;
        v.z = isSet(elem.z) ? 1 : 0;
        return v;
    }

    __device__ __host__ CTbyte3 GetNeutral(void)
    {
        CTbyte3 v = {0};
        return v;
    }
};

void ClipMask::ScanMasks(CTuint length)
{
//     for(CTbyte i = 0; i < 3; ++i)
//     {
//         maskScanner[i].ExcScan(mask[i].Begin(), mask[i].Begin() + length, ClipMaskPrefixSumOP());
//     }

    //mask3Scanner.ExcScan(mask3.Begin(), mask3.End(), ClipMaskPrefixSum3OP());
}

void EventLine::CompactClippedEvents(CTuint length)
{
//     PREPARE_KERNEL(length)
//         compactEventLine<<<grid, block>>>(GetDst(), GetSrc(), mask.Begin()(), eventScanner.GetPrefixSum().Begin()(), length);
//     }
}

void EventLine::ScanEventTypes(CTuint eventCount)
{
    EventStartScanOp<CTbyte> op0;
    CTbyte add = toggleIndex;
    typeStartScanner.ExcScan(type.Begin(add), type.Begin(add) + eventCount, op0);
}

void EventLines::BindToConstantMemory(hipStream_t pStream)
{
    cuEventLineTriple src;//(eventLines, 0);
    src.lines[0] = eventLines[0].GetPtr(0);
    src.lines[1] = eventLines[1].GetPtr(0);
    src.lines[2] = eventLines[2].GetPtr(0);


    cuEventLineTriple dst;//(eventLines, 1);
    dst.lines[0] = eventLines[0].GetPtr(1);
    dst.lines[1] = eventLines[1].GetPtr(1);
    dst.lines[2] = eventLines[2].GetPtr(1);

//     hipMemcpyToSymbol(HIP_SYMBOL(g_eventTriples), &src, sizeof(cuEventLineTriple));
//     hipMemcpyToSymbol(HIP_SYMBOL(g_eventTriples), &dst, sizeof(cuEventLineTriple), sizeof(cuEventLineTriple));

    hipMemcpyToSymbolAsync(HIP_SYMBOL(g_eventTriples), &src, sizeof(cuEventLineTriple), 0, hipMemcpyHostToDevice, pStream);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(g_eventTriples), &dst, sizeof(cuEventLineTriple), sizeof(cuEventLineTriple), hipMemcpyHostToDevice, pStream);


//     cuConstEventLineTriple constSrc;//(eventLines, 0);
//     src.lines[0] = eventLines[0].GetPtr(0);
//     src.lines[1] = eventLines[1].GetPtr(0);
//     src.lines[2] = eventLines[2].GetPtr(0);
// 
//     hipMemcpyToSymbolAsync(HIP_SYMBOL(g_eventSrcTriples), &constSrc, sizeof(cuConstEventLineTriple), 0, hipMemcpyHostToDevice);
//     hipMemcpyToSymbolAsync(HIP_SYMBOL(g_eventDstTriples), &dst, sizeof(cuEventLineTriple), 0, hipMemcpyHostToDevice);
}

// void EventLines::BindToggleIndexToConstantMemory(void)
// {
//     CTbyte dst = ((toggleIndex+1)%2);
//     hipMemcpyToSymbol(HIP_SYMBOL(g_eventSrcIndex), &toggleIndex, sizeof(CTbyte));
//     hipMemcpyToSymbol(HIP_SYMBOL(g_eventDstIndex), &dst, sizeof(CTbyte));
// }

void cuKDTreeScan::ScanClipMaskTriples(CTuint eventCount)
{
    ConstTuple<3, CTbyte> ptr;
    ptr.ts[0] = m_clipsMask.mask[0].GetConstPointer();
    ptr.ts[1] = m_clipsMask.mask[1].GetConstPointer();
    ptr.ts[2] = m_clipsMask.mask[2].GetConstPointer();

    Tuple<3, CTuint> ptr1;
    ptr1.ts[0] = m_clipsMask.scannedMask[0].GetPointer();
    ptr1.ts[1] = m_clipsMask.scannedMask[1].GetPointer();
    ptr1.ts[2] = m_clipsMask.scannedMask[2].GetPointer();
    
    Tuple<3, CTuint> sums;
    sums.ts[0] = m_clipsMask.scannedSums[0].GetPointer();
    sums.ts[1] = m_clipsMask.scannedSums[1].GetPointer();
    sums.ts[2] = m_clipsMask.scannedSums[2].GetPointer();

    ClipMaskPrefixSumOP op;
    ScanBinaryTriples(ptr, ptr1, sums, eventCount, op, m_pStream);

    //m_clipsMask.maskScanner[0].ExcScan(m_clipsMask.mask[0].Begin(), m_clipsMask.mask[0].Begin() + eventCount, op, m_pStream);
//     m_clipsMask.maskScanner[1].ExcScan(m_clipsMask.mask[1].Begin(), m_clipsMask.mask[1].Begin() + eventCount, op, m_pStream);
//     m_clipsMask.maskScanner[2].ExcScan(m_clipsMask.mask[2].Begin(), m_clipsMask.mask[2].Begin() + eventCount, op, m_pStream);

}

CT_RESULT cuKDTreeScan::Update(void)
{
    if(!m_initialized)
    {
        InitBuffer();
        m_initialized = true;
    }

    //ClearBuffer();
    
    CTuint primitiveCount = m_currentTransformedVertices.Size() / 3;

    static bool staticc = true;

    cudaCreateTriangleAABBs(m_currentTransformedVertices.GetPointer(), m_primAABBs.GetPointer(), primitiveCount, m_pStream);

   // if(staticc)
    {
        DEVICE_SYNC_CHECK();

        static float3 max3f = {FLT_MAX, FLT_MAX, FLT_MAX};
        static float3 min3f = -max3f;

        BBox bboxN;
        bboxN.m_min = max3f; 
        bboxN.m_max = min3f;
        m_sceneBBox.Resize(m_primAABBs.Size()/2);
        nutty::Reduce(m_sceneBBox.Begin(), m_primAABBs.Begin(), m_primAABBs.End(), ReduceBBox(), bboxN, m_pStream);
        staticc = false;
    }

    DEVICE_SYNC_CHECK(); 
    
    CTuint elementBlock = EVENT_GROUP_SIZE;//nutty::cuda::GetCudaBlock(primitiveCount, 256U);
    CTuint elementGrid = nutty::cuda::GetCudaGrid(primitiveCount, elementBlock);
    
    m_eventLines.Resize(2 * primitiveCount, m_pStream);

#ifdef PROFILE
        chimera::util::HTimer g_timer;
        hipDeviceSynchronize();
        g_timer.Start();
        g_time = 0;
#endif

    m_eventLines.toggleIndex = 0;

    createEventsAndInit3<1, 0><<<elementGrid, elementBlock, 0, m_pStream>>>(
        m_primAABBs.GetConstPointer(), 
        m_sceneBBox.GetConstPointer(),

        m_activeNodes.GetPointer(),
        m_nodes_NodeIdToLeafIndex.GetPointer(),
        m_nodes_IsLeaf.GetPointer(),
        m_nodes_ContentCount.GetPointer(),
        m_nodesBBox[0].GetPointer(),

        primitiveCount);

    DEVICE_SYNC_CHECK();

    for(CTbyte i = 0; i < 3; ++i)
    {
        nutty::Sort(
            nutty::DevicePtr_Cast<IndexedEvent>(m_eventLines.eventLines[i].GetPtr(0).indexedEvent), 
            nutty::DevicePtr_Cast<IndexedEvent>(m_eventLines.eventLines[i].GetPtr(0).indexedEvent + 2 * primitiveCount), 
            EventSort(),
            m_pStream);
    }
    DEVICE_SYNC_CHECK();

    reorderEvent3<<<2 * elementGrid, elementBlock, 0, m_pStream>>>(2 * primitiveCount);

    DEVICE_SYNC_CHECK();

    CTuint g_interiorNodesCountOnThisLevel = 1;
    CTuint g_currentInteriorNodesCount = 1;
    CTuint g_currentLeafCount = 0;
    CTuint g_leafContentOffset = 0;
    CTuint g_childNodeOffset = 1;
    CTuint g_childNodeOffset2 = 1;
    CTuint g_nodeOffset = 0;
    CTuint g_nodeOffset2 = 0;
    CTuint g_lastChildCount = 0;
    CTuint g_entries = 1;
    
    CTuint eventCount = 2 * primitiveCount;

    m_eventLines.Toggle();
    
    CTuint maxDepth = 0;

    for(CTbyte d = 0; d <= m_depth; ++d)
    {
        static int i = 0;
        //__ct_printf("New Level=%d Events=%d (Frame=%d)\n", d, eventCount, ++i);
        
        CTuint nodeCount = g_interiorNodesCountOnThisLevel;
        CTuint nodeBlock = NODES_GROUP_SIZE;//nutty::cuda::GetCudaBlock(nodeCount, 256U);
        CTuint nodeGrid = nutty::cuda::GetCudaGrid(nodeCount, nodeBlock);

        CTuint eventBlock = EVENT_GROUP_SIZE;//nutty::cuda::GetCudaBlock(eventCount, 256U);
        CTuint eventGrid = nutty::cuda::GetCudaGrid(eventCount, eventBlock);

        DEVICE_SYNC_CHECK();

#if defined _DEBUG
#if defined DYNAMIC_PARALLELISM
        //if(nodeCount < 0) //6
        {
#endif
            //m_hNodesContentCount.Resize(nodeCount);
            m_dthAsyncNodesContent.Resize(nodeCount);
            m_dthAsyncNodesContent.StartCopy(m_nodes_ContentCount.GetConstPointer(), 0, nodeCount);
            //nutty::Copy(m_hNodesContentCount.Begin(), m_nodes_ContentCount.Begin(), nodeCount);

#if defined DYNAMIC_PARALLELISM
        }
#endif
#endif

#if 0
        m_hNodesContentCount.Resize(nodeCount);
        nutty::Copy(m_hNodesContentCount.Begin(), m_nodes_ContentCount.Begin(), nodeCount);
     //   PrintBuffer(m_hNodesContentCount, nodeCount);
//         for(int i = 0; i < nodeCount; ++i)
//         {
//             if(m_hNodesContentCount[i] > 500000 || m_hNodesContentCount[i] <= MAX_ELEMENTS_PER_LEAF)
//             {
//                 exit(0);
//             }
//         }

        //PrintBuffer(m_nodes_ContentCount, nodeCount);

        PRINT_BUFFER_N(m_nodes_ContentCount, nodeCount);
#endif

        //m_pool.ClearEvents();

        ComputeSAH_Splits(
            nodeCount, 
            eventCount,
            m_nodes_ContentCount.Begin()());

        DEVICE_SYNC_CHECK();

        makeLeafIfBadSplitOrLessThanMaxElements<<<nodeGrid, nodeBlock, 0, m_pStream>>>(
            m_nodes,
            m_nodes_IsLeaf.GetPointer() + g_nodeOffset,
            m_activeNodes.GetPointer(),
            m_activeNodesIsLeaf.GetPointer(), 
            m_splits,
            d == m_depth-1,
            nodeCount);

        DEVICE_SYNC_CHECK();

        m_newNodesContentCount.Resize(m_nodes_ContentCount.Size());
        m_newNodesContentStartAdd.Resize(m_nodes_ContentCount.Size());

        m_lastNodeContentStartAdd.Resize(m_newNodesContentStartAdd.Size());

        CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_lastNodeContentStartAdd.GetPointer(), m_nodes_ContentStartAdd.GetPointer(), nodeCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));

        MakeLeavesResult leavesRes; // = MakeLeaves(m_activeNodesIsLeaf.Begin(), g_nodeOffset, 0, nodeCount, eventCount, g_currentLeafCount, g_leafContentOffset, 0);
        leavesRes.leafCount = 0;
        leavesRes.interiorPrimitiveCount = eventCount/2;
        leavesRes.leafPrimitiveCount = 0;

        CTuint lastLeaves = leavesRes.leafCount;
        primitiveCount = leavesRes.interiorPrimitiveCount;

        if(leavesRes.leafCount) //assert(!leavesRes.leafCount && "currently not working");
        {
            OutputDebugStringA("leavesRes.leafCount currently not working\n");
            exit(0);
        }
        
        DEVICE_SYNC_CHECK();
           
        CTuint count = eventCount;
        
        if(leavesRes.interiorPrimitiveCount)
        {
            CTuint block = EVENT_GROUP_SIZE;
            CTuint grid = nutty::cuda::GetCudaGrid(count, block);    

            m_eventLines.Resize(2 * count, m_pStream);
            m_clipsMask.Resize(2 * count, m_pStream);

//             nutty::ZeroMem(m_clipsMask.mask[0]);
//             nutty::ZeroMem(m_clipsMask.mask[1]);
//             nutty::ZeroMem(m_clipsMask.mask[2]);

//             nutty::ZeroMem(m_clipsMask.maskScanner[0].GetPrefixSum());
//             nutty::ZeroMem(m_clipsMask.maskScanner[1].GetPrefixSum());
//             nutty::ZeroMem(m_clipsMask.maskScanner[2].GetPrefixSum());

//              CTuint tb = 32;
//              CTuint tg = nutty::cuda::GetCudaGrid(count, tb);

            createClipMask<<<grid, block, 0, m_pStream>>>(
                m_nodes_ContentStartAdd.GetPointer(), 
                m_nodes_ContentCount.GetPointer(),
                count,
                m_eventLines.toggleIndex);

//             clipEvents3<<<grid, block, 0, m_pStream>>>(
//                 m_nodes_ContentStartAdd.GetPointer(), 
//                 m_nodes_ContentCount.GetPointer(),
//                 count,
//                 m_eventLines.toggleIndex);



            CTuint toggleSave = m_eventLines.toggleIndex;

// CTuint prefixSums[3];
// for(int k = 0; k < 3; ++k)
// {
//     nutty::HostBuffer<CTuint> srcEventScan(2 * count);
//     nutty::Copy(srcEventScan.Begin(), m_clipsMask.mask[k].Begin(), m_clipsMask.mask[k].Begin() + 2 * count);
//     prefixSums[k] = 0;
//     for(int i = 0; i < srcEventScan.Size(); ++i)
//     {
//         prefixSums[k] += srcEventScan[i] > 0;
//     }
// }
            DEVICE_SYNC_CHECK();

            //m_clipsMask.ScanMasks(2 * count);
            ScanClipMaskTriples(2 * count);
            //m_clipsMask.mask3Scanner.ExcScan(m_clipsMask.mask3.Begin(), m_clipsMask.mask3.Begin() + 2 * count, ClipMaskPrefixSum3OP());
// 
            m_dthAsyncIntCopy.WaitForStream(m_stream);
            m_dthAsyncByteCopy.WaitForStream(m_stream);

            //m_dthAsyncIntCopy.StartCopy((CTuint*)(m_clipsMask.scannedMask[0].GetConstPointer() + count - 1), 0);
            m_dthAsyncIntCopy.StartCopy((CTuint*)(m_clipsMask.scannedMask[0].GetConstPointer() + 2 * count - 1), 1);

            //m_dthAsyncByteCopy.StartCopy((CTbyte*)(m_clipsMask.mask[0].GetPointer() + count - 1), 0);
            m_dthAsyncByteCopy.StartCopy((CTbyte*)(m_clipsMask.mask[0].GetPointer() + 2 * count - 1), 1);

            CTuint _block = EVENT_GROUP_SIZE;
            CTuint _grid = nutty::cuda::GetCudaGrid(2 * count, block);

            compactEventLineV2<<<_grid, _block, 0, m_pStream>>>(
                2 * count,
                m_eventLines.toggleIndex);
            
            DEVICE_SYNC_CHECK();
            
            m_eventLines.Toggle();

            g_leafContentOffset += leavesRes.leafPrimitiveCount;

            if(lastLeaves)
            {
                setActiveNodesMask<1><<<nodeGrid, nodeBlock, 0, m_pStream>>>(
                    m_activeNodesThisLevel.Begin()(), 
                    m_activeNodesIsLeaf.Begin()(), 
                    m_interiorCountScanned.Begin()(),
                    0, 
                    nodeCount);
            }
            
            CTuint childCount = (nodeCount - leavesRes.leafCount) * 2;
            CTuint thisLevelNodesLeft = nodeCount - leavesRes.leafCount;
            
            nodeBlock = NODES_GROUP_SIZE;
            nodeGrid = nutty::cuda::GetCudaGrid(thisLevelNodesLeft, nodeBlock);

            initInteriorNodes<<<nodeGrid, nodeBlock, 0, m_pStream>>>(
                m_activeNodes.GetConstPointer(),
                m_activeNodesThisLevel.GetConstPointer(),

                m_nodesBBox[0].GetConstPointer(), 
                m_nodesBBox[1].GetPointer(), 

                m_nodes_ContentCount.GetPointer(),

                m_newNodesContentCount.GetPointer(),
                m_newActiveNodes.GetPointer(),
                m_activeNodesIsLeaf.GetPointer() + nodeCount,

                g_childNodeOffset,
                g_nodeOffset,
                thisLevelNodesLeft,
                m_lastNodeContentStartAdd.GetPointer(),
                m_gotLeaves.GetPointer(),
                m_depth == d+1,
                leavesRes.leafCount);

            CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_activeNodes.GetPointer(), m_newActiveNodes.GetPointer(), childCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));
            CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_nodes_ContentCount.GetPointer(), m_newNodesContentCount.GetPointer(), childCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));
            //CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_nodes_ContentStartAdd.GetPointer(), m_newNodesContentStartAdd.GetPointer(), childCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));

            DEVICE_SYNC_CHECK();

            m_dthAsyncIntCopy.WaitForCopy();
            m_dthAsyncByteCopy.WaitForCopy();
 
            //CTuint ccLeft = m_dthAsyncIntCopy[0] + isSet(m_dthAsyncByteCopy[0]);

            eventCount = m_dthAsyncIntCopy[1] + isSet(m_dthAsyncByteCopy[1]);// - ccLeft; //m_clipsMask.scannedMask[0][2 * count - 1] + isSet(m_clipsMask.mask[0][2 * count - 1]);

            if(eventCount == 0)
            {
                 __ct_printf("FATAL ERROR eventCount %d \n");
                exit(0);
            }

//             PrintBuffer(m_eventLines.eventLines[0].nodeIndex->Get(m_eventLines.toggleIndex), eventCount);
//             PrintBuffer(m_eventLines.eventLines[0].mask, 2 * count);
//             PrintBuffer(m_clipsMask.scannedMask[0], 2 * count);

            m_dthAsyncByteCopy.WaitForStream(m_stream);
            m_dthAsyncByteCopy.StartCopy(m_gotLeaves.GetConstPointer(), 0);
           
            eventBlock = EVENT_GROUP_SIZE;
            eventGrid = nutty::cuda::GetCudaGrid(eventCount, eventBlock);
            
            setEventsBelongToLeafAndSetNodeIndex<<<eventGrid, eventBlock, 0, m_pStream>>>(
                m_activeNodesIsLeaf.GetPointer() + nodeCount,
                m_eventIsLeaf.GetPointer(),
                m_nodes_NodeIdToLeafIndex.GetPointer() + g_childNodeOffset,
                eventCount,
                2 * nodeCount,
                m_eventLines.toggleIndex);

            DEVICE_SYNC_CHECK();

            //PROFILE_END;

            //if(!m_dthAsyncByteCopy[0])
            {
                m_interiorContentScanner.Resize(childCount);
                m_interiorContentScanner.ExcScan(m_nodes_ContentCount.Begin(), m_nodes_ContentCount.Begin() + childCount, nutty::PrefixSumOp<CTuint>());
                
                DEVICE_SYNC_CHECK();

                CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(
                    m_nodes_ContentStartAdd.GetPointer(), m_interiorContentScanner.GetPrefixSum().GetConstPointer(), childCount * sizeof(CTuint), hipMemcpyDeviceToDevice, m_pStream));
                //nutty::Copy(m_nodes_ContentStartAdd.Begin(), m_interiorContentScanner.GetPrefixSum().Begin(), m_interiorContentScanner.GetPrefixSum().Begin() + childCount);

                DEVICE_SYNC_CHECK();    
            }

            m_dthAsyncByteCopy.WaitForCopy();

            DEVICE_SYNC_CHECK();
#if 0
            if(eventCount != 2 * (m_nodes_ContentCount[childCount - 1] + m_nodes_ContentStartAdd[childCount - 1]))
            {
                CTuint sum = 0;
                CTuint oldSum = 0;
                CTuint offset = 0;
                CTuint oldContentCount = 0;
                __ct_printf("nodeCount=%d\n", nodeCount);
                PrintBuffer(m_hNodesContentCount, nodeCount);
                PrintBuffer(m_lastNodeContentStartAdd, nodeCount);
                for(int i = 0; i < nodeCount; ++i)
                {
                    CTuint elemsLeft = 4 * m_lastNodeContentStartAdd[i];
                    CTuint contentCount = m_hNodesContentCount[i];
                    CTuint add = m_splits_IndexedSplit[oldSum].index;
                    CTbyte axis = m_splits_Axis[add];
                    CTuint above = m_splits_Above[add];
                    CTuint below = m_splits_Below[add];
                    oldSum += 2 * m_hNodesContentCount[i];

//                     if(IS_INVALD_SAH(m_splits_IndexedSplit[oldSum].sah))
//                     {
//                         __ct_printf("Invalid sah...\n");
//                     }
 
// 
                    //offset += 2 * below;
                    //aboveSum += 2 * above;
//                     //PrintBuffer(m_clipsMask.scannedMask[axis], 2 * oldContentLength + 2 * oldContentLength);
//                     //__ct_printf("\n");
//                     //PrintBuffer(m_clipsMask.mask[axis], 2 * oldContentLength + 2 * oldContentLength);
// 
//                     for(int i = 0; i < 2 * count - 1; ++i)
//                     {
//                         scannedMaskFile << m_clipsMask.scannedMask[axis][i] << "\n";
//                         maskFile << m_clipsMask.mask[axis][i] << "\n";
//                     }
// 
//                     scannedMaskFile.close();
//                     maskFile.close();
                    
                    //__ct_printf("\n%d %d %d\n", oldContentLength, 2 * oldContentLength + 2 * oldContentLength);
                    CTuint m0 = m_clipsMask.mask[axis][elemsLeft + 2 * contentCount - 1];
                    CTuint m1 = m_clipsMask.mask[axis][elemsLeft + 4 * contentCount - 1];

                    CTuint first = m_clipsMask.scannedMask[axis][elemsLeft + 2 * contentCount - 1];
                    CTuint second = m_clipsMask.scannedMask[axis][elemsLeft + 4 * contentCount - 1];

                    CTuint eventsLeft = first + (m0 > 0 ? 1 : 0) - m_clipsMask.scannedMask[axis][elemsLeft];

                    CTuint eventsRight = second + (m1 > 0 ? 1 : 0) - eventsLeft - m_clipsMask.scannedMask[axis][elemsLeft];

                    CTuint mask1 = 0; CTuint mask = 0;
//                     for(int i = elemsLeft; i < elemsLeft + 2 * contentCount; ++i)
//                     {
//                         mask1 += (m_clipsMask.mask[axis][i] != 0 ? 1 : 0);
// //                              scannedMaskFile << m_clipsMask.scannedMask[axis][i] << "\n";
// //                              maskFile << (m_clipsMask.mask[axis][i] != 0 ? 1 : 0) << "\n";
//                     }
// 

//                     for(int i = elemsLeft + 2 * contentCount; i < elemsLeft + 4 * contentCount; ++i)
//                     {
//                         mask += (m_clipsMask.mask[axis][i] != 0 ? 1 : 0);
// //                              scannedMaskFile << m_clipsMask.scannedMask[axis][i] << "\n";
// //                              maskFile << (m_clipsMask.mask[axis][i] != 0 ? 1 : 0) << "\n";
//                     }

                    __ct_printf("%d %d - %d %d - %d %d masks=%d %d, scan=%d elemsLeft=%d\n", 
                        2 * below, 2 * above, eventsLeft, eventsRight, mask, mask1, m0, m1, m_clipsMask.scannedMask[axis][2 * count-1], elemsLeft);
// 
// 
//                     oldContentCount = offset;
// 
                    // offset += 2 * below + 2 * above;
//                     std::ofstream maskFile("nodeMaskCreation.txt");
//                     CTuint ec = 0;
//                 CTuint srcAdd = toggleSave;
// 
//                     nutty::HostBuffer<IndexedSAHSplit> hisplit(count); nutty::Copy(hisplit.Begin(), m_splits_IndexedSplit.Begin(), m_splits_IndexedSplit.Begin() + count);
// 
//                     nutty::HostBuffer<CTuint> hContentStart(count); nutty::Copy(hContentStart.Begin(), m_lastNodeContentStartAdd.Begin(), m_lastNodeContentStartAdd.Begin() + count);
// 
//                     nutty::HostBuffer<CTbyte> hsplitAxis(count); nutty::Copy(hsplitAxis.Begin(), m_splits_Axis.Begin(), m_splits_Axis.Begin() + count);
// 
//                     nutty::HostBuffer<CTreal> hSplit(count); nutty::Copy(hSplit.Begin(), m_splits_Plane.Begin(), m_splits_Plane.Begin() + count);
// 
//                     nutty::HostBuffer<BBox> hbbox(count); 
//                     nutty::Copy(hbbox.Begin(), m_eventLines.eventLines[axis].ranges[srcAdd].Begin(), m_eventLines.eventLines[axis].ranges[srcAdd].Begin() + count);
// 
//                     nutty::HostBuffer<IndexedEvent> hv(count); 
//                     nutty::Copy(hv.Begin(), m_eventLines.eventLines[axis].indexedEvent[srcAdd].Begin(), m_eventLines.eventLines[axis].indexedEvent[srcAdd].Begin() + count);
// 
//                     nutty::HostBuffer<CTbyte> htype(count); nutty::Copy(htype.Begin(), m_eventLines.eventLines[axis].type[srcAdd].Begin(), m_eventLines.eventLines[axis].type[srcAdd].Begin() + count);
// 
//                     for(int id = 0; id < count; ++id)
//                     {
//               
//                         CTuint nodeIndex = 0;//m_eventLines.eventLines[axis].nodeIndex->Get(srcAdd)[id];
//                         CTuint eventsLeftFromMe = 2 * hContentStart[nodeIndex];
//                         IndexedSAHSplit isplit = hisplit[eventsLeftFromMe];
//                         CTbyte splitAxis = hsplitAxis[isplit.index];
//                         CTreal split = hSplit[isplit.index];
//                         CTuint N = eventsLeftFromMe + 2 * m_hNodesContentCount[nodeIndex];
//   
//                         BBox bbox = hbbox[id];
//                         CTreal v = hv[id].v;    
//                         CTbyte type = htype[id];
// 
//                         CTreal minAxis = getAxis(bbox.m_min, splitAxis);
//                         CTreal maxAxis = getAxis(bbox.m_max, splitAxis);
//                         
//                         if(maxAxis <= split)
//                         {
//                             maskFile << "LEFT index=";
//                             ec++;
//                         }
//                         else if(minAxis >= split)
//                         {
//                             maskFile << "RIGHT index=";
//                             ec++;
//                         }
//                         else
//                         {
//                             maskFile << "CLIP index=";
//                             ec += 2;
//                         }
// 
//                         maskFile << isplit.index << " maxAxis=" << maxAxis << " minAxis=" << minAxis << " split=" << split << " v=" << v << " nodeIndex=" << nodeIndex << " splitAxis=" << splitAxis << " axis=" << axis << "\n";
//                     }
//                     maskFile.close();
// 
//                     __ct_printf("testcount=%d\n", ec);
                }

                __ct_printf("%d %d\n", d, childCount);

                __ct_printf("FATAL ERROR eventCount %d != %d\n", eventCount, 2 * (m_nodes_ContentCount[childCount - 1] + m_nodes_ContentStartAdd[childCount - 1]));

                __debugbreak();
            }
#endif
            leavesRes = MakeLeaves(
                m_activeNodesIsLeaf.Begin(),
                g_childNodeOffset, 
                nodeCount, 
                childCount,
                eventCount, 
                g_currentLeafCount + lastLeaves, 
                g_leafContentOffset, 1, 
                m_dthAsyncByteCopy[0]);

            DEVICE_SYNC_CHECK();

            const static CTbyte null = 0;
            CUDA_RT_SAFE_CALLING_SYNC(hipMemcpyAsync(m_gotLeaves.GetPointer(), &null, sizeof(CTbyte), hipMemcpyHostToDevice, m_pStream));
            
            DEVICE_SYNC_CHECK();

            eventCount = 2 * leavesRes.interiorPrimitiveCount;
        }
        else
        {
            //todo
            for(int i = 0; i < nodeCount; ++i)
            {
                m_nodes_IsLeaf.Insert(g_nodeOffset + i, (CTbyte)1);
            }

            __ct_printf("errr not good...\n");
        }

        g_entries += 2 * nodeCount;
        g_lastChildCount = 2 * nodeCount;
        g_nodeOffset2 = g_nodeOffset;
        g_interiorNodesCountOnThisLevel = 2 * (nodeCount - lastLeaves) - leavesRes.leafCount;
        g_currentInteriorNodesCount += g_interiorNodesCountOnThisLevel;
        g_nodeOffset = g_childNodeOffset;
        g_childNodeOffset += 2 * (nodeCount);

        //update globals
        g_leafContentOffset += leavesRes.leafPrimitiveCount;
        
        g_currentLeafCount += lastLeaves + leavesRes.leafCount;

        ct_printf(
            "g_nodeOffset=%d g_childNodeOffset=%d g_leafContentOffset=%d g_interiorNodesCountOnThisLevel=%d g_currentInteriorNodesCount=%d g_currentLeafCount=%d\nCreated '%d' Leaves, Interior Nodes '%d'\n", 
              g_nodeOffset, g_childNodeOffset, g_leafContentOffset, g_interiorNodesCountOnThisLevel, g_currentInteriorNodesCount, g_currentLeafCount, lastLeaves + leavesRes.leafCount, g_interiorNodesCountOnThisLevel);

        DEVICE_SYNC_CHECK();

        if(!leavesRes.leafCount)
        {
            hipMemcpyAsync(m_nodesBBox[0].GetPointer(), m_nodesBBox[1].GetConstPointer(), g_interiorNodesCountOnThisLevel * sizeof(BBox), hipMemcpyDeviceToDevice, m_pStream);
            //nutty::Copy(m_nodesBBox[0].Begin(), m_nodesBBox[1].Begin(), m_nodesBBox[1].Begin() + g_interiorNodesCountOnThisLevel);
        }

        maxDepth = d;
        if(eventCount == 0 || g_interiorNodesCountOnThisLevel == 0) //all nodes are leaf nodes
        {
            //primitiveCount = lastCnt;
            break;
        }

        if(d < m_depth-1) //are we not done?
        {
            //check if we need more memory
            if(eventCount > m_splits_Above.Size())
            {
                GrowSplitMemory(2 * eventCount);
            }

            if(m_activeNodes.Size() < g_interiorNodesCountOnThisLevel + 2 * g_interiorNodesCountOnThisLevel)
            {
                GrowPerLevelNodeMemory(4 * 2 * g_interiorNodesCountOnThisLevel);
            }

            if(m_nodes_IsLeaf.Size() < (g_childNodeOffset + 2 * g_interiorNodesCountOnThisLevel))
            {
                GrowNodeMemory();
            }
        }
    }

#ifdef PROFILE
    hipDeviceSynchronize();
    g_timer.Stop();
    __ct_printf("Total: %f, Section: %f (maxdepth=%d)\n", g_timer.GetMillis(), g_time, maxDepth);
#endif

    m_interiorNodesCount = g_currentInteriorNodesCount;
    m_leafNodesCount = g_currentLeafCount;
    CTuint allNodeCount = m_interiorNodesCount + m_leafNodesCount;

#ifdef _DEBUG
    ValidateTree();
#endif

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipStreamSynchronize(m_pStream));

//     ct_printf("Tree Summary:\n");
//     PRINT_BUFFER(m_nodes_IsLeaf);
//     PRINT_BUFFER(m_nodes_Split);
//     PRINT_BUFFER(m_nodes_SplitAxis);
//     PRINT_BUFFER(m_nodes_LeftChild);
//     PRINT_BUFFER(m_nodes_RightChild);
//     PRINT_BUFFER(m_leafNodesContentCount);
//     PRINT_BUFFER(m_leafNodesContentStart);
//     PRINT_BUFFER(m_nodes_NodeIdToLeafIndex);

    if(m_leafNodesContent.Size() < 1024)
    {
        PRINT_BUFFER(m_leafNodesContent);
    }
    else
    {
        ct_printf("skipping content '%d' elements...\n", m_leafNodesContent.Size());
    }

    DEVICE_SYNC_CHECK();

    return CT_SUCCESS;
}

void cuKDTreeScan::ValidateTree(void)
{
    std::queue<CTuint> queue;

    queue.push(0);

    while(!queue.empty())
    {
        CTuint node = queue.front();
        queue.pop();
        ct_printf("%d ", node);
        if(!m_nodes_IsLeaf[node])
        {
            ct_printf("\n");
             //assertions are happening here if we are out of bounds
            CTuint left = m_nodes_LeftChild[node];
            CTuint right = m_nodes_RightChild[node];
            if(left < node || right < node)
            {
                assert(0 && "fuck");
            }
            queue.push(left);
            queue.push(right);
        }
        else
        {
            CTuint leafIndex = m_nodes_NodeIdToLeafIndex[node];
            ct_printf(" - %d\n", leafIndex);
        }
    }
}