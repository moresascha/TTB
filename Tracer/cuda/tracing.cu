#include "hip/hip_runtime.h"
#ifdef _DEBUG
#define NUTTY_DEBUG
#endif

#include <cutil_math.h>
#include "globals.cuh"
#include <Nutty.h>
#include <DeviceBuffer.h>
#include <Scan.h>
#include <Copy.h>
#include <Functions.h>
#include <Reduce.h>
#include <Fill.h>
#include <cuda/cuda_helper.h>
#include "../print.h"
#include "../double_math.h"
#include "../texture_array.h"
#include "lighting.cuh"
#include <list>
#include <queue>
#include <map>
#include "geometry.cuh"
#include "traversing.cuh"
#include "vector_functions.cuh"
#include "culling.cuh"
#include "material.cuh"
#include "texturing.cuh"
#include "shading.cuh"
#include "tracer_api.cuh"
#include "post_processing.cuh"

#undef MEASURING

struct SurfaceObject
{
    hipArray* cuArray;
    hipSurfaceObject_t surfObj;
    SurfaceObject(void)
    {

    }

    void Init(uint width, uint height)
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); 
        hipMallocArray(&cuArray, &channelDesc, width, height, hipArraySurfaceLoadStore); 
        hipResourceDesc resDesc; 
        memset(&resDesc, 0, sizeof(resDesc)); 
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;        
        hipCreateSurfaceObject(&surfObj, &resDesc);

       // hipMemcpyToArray(cuInputArray, 0, 0, h_data, size, hipMemcpyHostToDevice);
    }

    ~SurfaceObject(void)
    {
        hipDestroySurfaceObject(surfObj);

        hipFreeArray(cuArray);
    }
};

struct RWTextureObject
{
    hipTextureObject_t tex;
    hipArray_t cuArray;
    hipSurfaceObject_t surfObj;

    void Init(uint width, uint height)
    {
        hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();

        CUDA_RT_SAFE_CALLING_NO_SYNC(hipMallocArray(&cuArray, &desc, width, height));

        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        memset(&resDesc.res, 0, sizeof(resDesc.res));
        resDesc.res.array.array = cuArray;

        hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.addressMode[0] = hipAddressModeWrap;
        texDesc.addressMode[1] = hipAddressModeWrap;
        texDesc.addressMode[2] = hipAddressModeWrap;
        texDesc.filterMode = hipFilterModeLinear;
        texDesc.normalizedCoords = 1;
       // texDesc.readMode = hipReadModeNormalizedFloat;

        CUDA_RT_SAFE_CALLING_NO_SYNC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));
      
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipCreateSurfaceObject(&surfObj, &resDesc));
    }

    ~RWTextureObject(void)
    {
        hipDestroyTextureObject(tex);
        hipFreeArray(cuArray);
        hipDestroySurfaceObject(surfObj);
    }
};

__constant__ Real3 g_sunPos;

__constant__ float3 g_view[3];

__constant__ BBox g_bbox;

__constant__ TreeNodes g_tree;

__constant__ Triangles g_geometry;

__device__ uint* debugInfo = 0;

__constant__ float g_envMapScale;

__constant__ uint g_shader = 0;

static const int MAX_LIGHTS = 16;

__constant__ Light g_lights[MAX_LIGHTS];
__constant__ int g_currentLightCount;

__device__ int getCurrentShader(void)
{
    return g_shader;
}

__device__ int getLightCount(void)
{
    return g_currentLightCount;
}

__device__ const Light& getLight(int index)
{
    return g_lights[index];
}


__device__ float getGlobalIllum(void)
{
    return g_envMapScale;
}

struct rtLight : public _RT_Light
{
    int _index;
    float3 _position;
    float3 _intensity_N_radius;
    float3 _color;

    rtLight(int index) : _index(index)
    {
        _position.x = 0;
        _position.y = 6;
        _position.x = 0;
        _intensity_N_radius.x = 1;
        _intensity_N_radius.y = 1;
        _color.x = 1;
        _color.y = 1;
        _color.z = 1;
        Upload();
    }

    void Upload(void)
    {
        Light l;
        l.color = _color;
        l.intensity = _intensity_N_radius;
        l.position = _position;
        hipMemcpyToSymbolAsync(HIP_SYMBOL(g_lights), &l, sizeof(Light), _index * sizeof(Light), hipMemcpyHostToDevice);
    }

    void SetPosition(const float3& pos)
    {
        _position = pos;
        Upload();
    }

    void SetColor(const float3& c)
    {
        _color = c;
        Upload();
    }

    void SetIntensity(float in)
    {
        _intensity_N_radius.x = in;
        Upload();
    }

    void SetRadius(float r)
    {
        _intensity_N_radius.y = r;
        Upload();
    }

};

extern "C" __device__ const Triangles& getGeometry(void)
{
    return g_geometry;
}

extern "C" __device__ const BBox& getBBox(void)
{
    return g_bbox;
}

extern "C" __device__ Real3 getSunPos(void)
{
    return g_sunPos;
}

#define COMPUTE_SHADOW
#define COMPUTE_REFRACTION
#define RECURSION 1

#ifdef MEASURING
#define MAX_RECURSION 1
#else
#define MAX_RECURSION 5
#endif

#define RAY_WEIGHT_THRESHOLD 0.01

#define AIR_RI 1.00029
#define GLASS_RI 1.52

#define GLASS_TO_AIR (GLASS_RI / AIR_RI)
#define AIR_TO_GLASS (AIR_RI / GLASS_RI)

__device__ __forceinline int intersectP(const Real3& eye, const Real3& ray, const Real3& boxmin, const Real3& boxmax, Real* tmin, Real* tmax) 
{
    Real t0 = *tmin; Real t1 = *tmax;

    Real3 invRay = 1.0 / ray;

#pragma unroll 3
    for(byte i = 0; i < 3; ++i) 
    {
        float tNear = (getAxis(boxmin, i) - getAxis(eye, i)) * getAxis(invRay, i);
        float tFar = (getAxis(boxmax, i) - getAxis(eye, i)) * getAxis(invRay, i);

        if(tNear > tFar) 
        {
            float tmp = tNear;
            tNear = tFar;
            tFar = tmp;
        }

        t0 = tNear > t0 ? tNear : t0;
        t1 = tFar < t1 ? tFar : t1;

        if(t0 > t1) return 0;
    }
    *tmin = t0;
    *tmax = t1;
    return 1;
}

template <typename FC>
__device__  int hitsTriangle(const Real3& rayOrigin, Real min, Real max, const Real3& rayDirection, const Position* positions, Real* hitDepth, Real2& bary, FC cf, byte* isBackFace)
{
    Real3 p0 = positions[0];
    Real3 e1 = positions[1] - p0;
    Real3 e2 = positions[2] - p0;

    Real3 s1 = cross(rayDirection, e2);
    
    *isBackFace = (byte)(dot(cross(e1, e2), rayDirection) < 0);

    Real div = dot(e1, s1);
    if(div == 0) // || cf(e1, e2, rayDirection))
    {
        return 0;
    }

    Real invDiv = 1.0f / div;

    Real3 d = rayOrigin - p0;

    Real b1 = dot(d, s1) * invDiv;

    if(b1 < 0 || b1 > 1) 
    {
        return 0;
    }
        
    Real3 s2 = cross(d, e1);

    Real b2 = dot(rayDirection, s2) * invDiv;
        
    if(b2 < 0 || b2 + b1 > 1) 
    { 
        return false;
    }
        
    Real t = dot(e2, s2) * invDiv;
        
    if(t < min || t > max) 
    {
        return 0;
    }

    bary.x = b1;
    bary.y = b2;

    *hitDepth = t;

    return 1;
}

#define USE_SHORT_STACK

#ifdef USE_SHORT_STACK
struct ToDo 
{
    uint nodeIndex;
    Real tmax;
    Real tmin;
};
#endif

template <typename CF>
__device__ void traverse(const TreeNodes& n, const Real3& eye, const Real3& ray, TraceResult& hit, Real rayMin, Real rayMax, CF cf)
{ 
    hit.isHit = 0;

    Real tmin;
    Real tmax;
    tmin = tmax = rayMin;
    int pushDown = 1;
    uint nodeIndex = 0;

    uint root = 0;

#ifdef USE_SHORT_STACK
    int stackPos = 1;
    ToDo todo[16];
 
    todo[0].nodeIndex = 0;
    todo[0].tmin = rayMin;
    todo[0].tmax = rayMax;
#endif

    float t = FLT_MAX;

    while(tmax < rayMax) 
    {

#ifdef USE_SHORT_STACK
        if(stackPos == -1)
        {
            pushDown = 1;
            nodeIndex = root;
            tmin = tmax;
            tmax = rayMax;
        }
        else
        {
            stackPos--;
            nodeIndex = todo[stackPos].nodeIndex;
            tmin = todo[stackPos].tmin;
            tmax = todo[stackPos].tmax;
            pushDown = 0;
        }
#else
        uint nodeIndex = root;
        tmin = tmax;
        tmax = rayMax;
        pushDown = 1;
#endif

        while(!n.isLeaf[nodeIndex])
        {
            CTaxis_t axis = n.splitAxis[nodeIndex];

            Real nsplit = n.split[nodeIndex];

            Real tsplit = (nsplit - getAxis(eye, axis)) / getAxis(ray, axis);

            int belowFirst = (getAxis(eye, axis) < nsplit) || ((getAxis(eye, axis) == nsplit) && (getAxis(ray, axis) >= 0));

            uint first, second;

            if(belowFirst)
            {
                first = n.left(nodeIndex);
                second = n.right(nodeIndex);
            }
            else
            {
                first = n.right(nodeIndex);
                second = n.left(nodeIndex);
            }

            if(tsplit > tmax || tsplit <= 0)
            {
                nodeIndex = first;
            } 
            else if(tsplit <= tmin) 
            {
                nodeIndex = second;
            }
            else
            {
#ifdef USE_SHORT_STACK
                todo[stackPos].nodeIndex = second;
                todo[stackPos].tmin = tsplit;
                todo[stackPos].tmax = tmax;
                stackPos++;
#endif
                nodeIndex = first;
                tmax = tsplit;
                pushDown = 0;
            }

            if(pushDown)
            {
                root = nodeIndex;
            }
        }

        uint leafIndex = n.leafIndex[nodeIndex];
        uint start = n.contentStart[leafIndex];
        uint prims = n.contentCount[leafIndex];
        Real depth = rayMax;

        for(uint i = start; i < prims + start; ++i)
        {
            Real2 bary;
            Real d;
            uint triId = n.content[i];
            //debugInfo = (uint*)triId;
            //triId = 0;
            byte bf = 0;
            if(hitsTriangle(eye, tmin, rayMax, ray, getGeometry().positions + 3 * triId, &d, bary, cf, &bf))
            {
                if(d < depth)
                {
                    t = d;
                    hit.bary = bary;
                    hit.triIndex = triId;
                    hit.isBackFace = bf;
                    depth = d;
                    hit.isHit = 1;
                }
            }
        }

        if(hit.isHit && (t < tmax))
        {
            return;
        }
    }
}

struct RayPair
{
    Ray* rays;
    uint* mask;
};

struct RayRange
{
    size_t begin;
    size_t end;

    RayRange (void)
    {

    }

    RayRange(const RayRange& cpy)
    {
        begin = cpy.begin;
        end = cpy.end;
    }

    size_t Length()
    {
        return end - begin;
    }
};

__device__ Real3 transform3f(float3* m3x3l, const Real3* vector)
{
    return make_real3(dot(m3x3l[0], *vector), dot(m3x3l[1], *vector), dot(m3x3l[2], *vector));
}

__device__ Real3 transform4f(float4* m3x3l, const Real4* vector)
{
    return make_real3(dot(m3x3l[0], *vector), dot(m3x3l[1], *vector), dot(m3x3l[2], *vector));
}

__device__ void addRay(RayPair& pair, uint id, Ray& r)
{
    if(r.rayWeight > RAY_WEIGHT_THRESHOLD)
    {
        pair.mask[id] = 1;
        pair.rays[id] = r;
    }
}

__global__ void _traceShadowRays(float4* color, Ray* rays, unsigned int width, unsigned int height, unsigned int N)
{
    uint rayIndex = threadIdx.x + blockDim.x * blockIdx.x;

    if(rayIndex >= N)
    {
        return;
    }

    Ray r = rays[rayIndex];
    uint id = r.screenCoord.y * width + r.screenCoord.x;
    float4 c = color[id];

    Real3 d = getSunPos() - r.getOrigin();
    r.setDir(normalize(d));
    r.setMax(length(d));

    TraceResult hitRes;
    traverse(g_tree, r.getOrigin(), r.getDir(), hitRes, *r.getMin(), *r.getMax(), no_cull);

    if(hitRes.isHit)
    {
        Material mat = g_geometry.getMaterial(hitRes.triIndex);
        c *= (1 - r.rayWeight * mat.alpha());
        color[id] = c;
    }
}

__global__ void _traceRefractionRays(float4* color, Ray* refrRays, RayPair rays, unsigned int width, unsigned int height, unsigned int N)
{
    uint rayIndex = threadIdx.x + blockDim.x * blockIdx.x;

    if(rayIndex >= N)
    {
        return;
    }

     Ray r = refrRays[rayIndex];
     //r.clampToBBox();
     addRay(rays, rayIndex, r);

//     r.clampToBBox();
//     float3 os = r.getOrigin() + 0.1 * r.getDir();
//     r.origin_min.x = os.x;
//     r.origin_min.y = os.y;
//     r.origin_min.z = os.z;
//     addRay(rays, rayIndex, r);
//     color[r.screenCoord.y * width + r.screenCoord.x] = make_float4(1,0,0,0);

//     TraceResult hitRes;
//     traverse(g_tree, r.getOrigin(), r.getDir(), hitRes, *r.getMin(), *r.getMax(), cull_front);
// 
//     if(hitRes.isHit)
//     {
//         Real3 hitPos = g_geometry.getTrianglelHitPos(hitRes.triIndex, hitRes.bary);
//         Real3 normal = g_geometry.getTrianglelNormal(hitRes.triIndex, hitRes.bary);
//         
//         Material mat = g_geometry.getMaterial(hitRes.triIndex);
//         //if(
//         r.setOrigin(hitPos + RAY_HIT_NORMAL_DELTA * normal);
//         r.setDir(normalize(refract(r.getDir(), -normal, mat.reflectionIndex(), AIR_RI)));
//         r.clampToBBox();
//         addRay(rays, rayIndex, r);
// 
//         shade(hitRes, rayIndex, color, r, mat);
//     }
}

template <int WRITE_OUT>
__global__ void _traceRays(
    float4* color,
    Ray* inputRays,
    RayPair newReflectionRays,
    RayPair newRefractionRays,
    RayPair newShadowRays,
    unsigned int width,
    unsigned int height,
    unsigned int N)
{
    uint rayIndex = threadIdx.x + blockDim.x * blockIdx.x;

    if(rayIndex >= N)
    {
        return;
    }

    Ray r = inputRays[rayIndex];

    uint id = r.screenCoord.y * width + r.screenCoord.x;

    TraceResult hitRes;

    traverse(g_tree, r.getOrigin(), r.getDir(), hitRes, *r.getMin(), *r.getMax(), cull_back);

    if(hitRes.isHit)
    {
        Material mat = g_geometry.getMaterial(hitRes.triIndex);
        shade(hitRes, id, color, r, mat);
        //color[id].x = hitRes.isBackFace;
        Real3 hitPos = g_geometry.getTrianglelHitPos(hitRes.triIndex, hitRes.bary);
        Real3 normal = g_geometry.getTrianglelNormal(hitRes.triIndex, hitRes.bary);

        if(WRITE_OUT)
        {
            bool isTrans = mat.isTransp();
            bool isMirror = mat.isMirror();

            Real3 dir = r.getDir();
            Real weight = r.rayWeight;

            if(isTrans)
            {
                Real3 refraction;
                Real ratio;
                if(hitRes.isBackFace)
                {
                    refraction = refract(r.getDir(), -normal, mat.reflectionIndex(), AIR_RI);
                    ratio = Reflectance(r.getDir(), -normal, mat.reflectionIndex(), AIR_RI, mat.fresnel_t());
                }
                else
                {
                    refraction = refract(r.getDir(), normal, AIR_RI, mat.reflectionIndex());
                    ratio = Reflectance(r.getDir(), normal, AIR_RI, mat.reflectionIndex(), mat.fresnel_t());
                }

                if(abs(dot(refraction, refraction)) > 0)
                {
                    //Real ratio = Reflectance(r.getDir(), normal, AIR_RI, mat.reflectionIndex(), mat.fresnel_r());
                    Real3 reflection = reflect(r.getDir(), normal);
                    r.rayWeight = (1 - mat.alpha()) * ratio * weight * mat.reflectivity();// * (1 - mat.alpha()) * mat.reflectivity();
                    r.setDir(refraction);
                    r.setOrigin(hitPos + (hitRes.isBackFace ? +RAY_HIT_NORMAL_DELTA * normal : -RAY_HIT_NORMAL_DELTA * normal));
                    r.clampToBBox();
                    addRay(newRefractionRays, rayIndex, r);
                }
            }

            if(isMirror)
            {
                Real ratio = Reflectance(r.getDir(), normal, AIR_RI, mat.reflectionIndex(), mat.fresnel_r());
                Real3 reflection = reflect(r.getDir(), normal);
                r.rayWeight = ratio * weight * mat.reflectivity();
                r.setDir(reflection);
                r.setOrigin(hitPos + RAY_HIT_NORMAL_DELTA * normal);
                r.clampToBBox();
                addRay(newReflectionRays, rayIndex, r);
            }
        }
#if defined COMPUTE_SHADOW
        //spawn shadow ray forall lights
//         if(dot(normal, normalize(SUN_POS - hitPos)) > 0)
//         {
//             r.setOrigin(hitPos + RAY_HIT_NORMAL_DELTA * normal);
//             r.rayWeight = 0.5f;
//             addRay(newShadowRays, rayIndex, r);
//         }
        
        r.setOrigin(hitPos + RAY_HIT_NORMAL_DELTA * normal);
        r.rayWeight = 0;//0.5f;
        float4 c = color[id];
        for(int i = 0; i < getLightCount(); ++i)
        {
            const Light& light = getLight(i);
            float3 d = light.position - hitPos;
            float shadowScale = dot(normal, normalize(d));

            float distSquared = dot(d, d);

            if(distSquared < light.intensity.y * light.intensity.y)
            {
                if(shadowScale < 0)
                {
                    c *= (1 - 0.5 * -shadowScale);
                }
                else
                {
                    Real3 d = light.position - r.getOrigin();
                    r.setDir(normalize(d));
                    r.setMax(length(d));

                    TraceResult hitRes;
                    traverse(g_tree, r.getOrigin(), r.getDir(), hitRes, *r.getMin(), *r.getMax(), no_cull);

                    if(hitRes.isHit)
                    {
                        float in = fmaxf(0, 1-distSquared / (light.intensity.y * light.intensity.y));
                        Material mat = g_geometry.getMaterial(hitRes.triIndex);
                        c *= lerp(1.0f, 1 - 0.5 * shadowScale * in, mat.alpha());
                    }
                }
            }
        }
        color[id] = c;
#endif
    }
    else
    {
        float u = 0.5 + atan2(r.dir_max.z, r.dir_max.x) / (2 * PI);
        float v = 0.5 - asin(r.dir_max.y) / PI;
        float2 tc = make_float2(u, 1-v);
        float4 c  = readTexture(0, tc);
        color[id].x = color[id].x * (1 - r.rayWeight) + c.x;
        color[id].y = color[id].y * (1 - r.rayWeight) + c.y;
        color[id].z = color[id].z * (1 - r.rayWeight) + c.z;
    }
}

template<uint blockSize>
__global__ void _dpTraceRays(
    float4* color,
    Ray* rayMemory,
//     RayPair newReflectionRays,
//     RayPair newRefractionRays,
//     RayPair newShadowRays,
    float3 eye,
    unsigned int width,
    unsigned int height,
    unsigned int N)
{
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;
    uint idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint id = idx + idy * blockDim.x * gridDim.x;

    uint linearBlockThreadId = blockDim.x * threadIdx.y + threadIdx.x;
    uint linearBlock = gridDim.x * blockIdx.y + blockIdx.x;

    uint mask = 0;

    Ray r;

    color[id] = make_float4(0,0,0,0); //clear buffer

    if(!(idx >= width || idy >= height))
    {
        Real u = (Real)idx / (Real)width;
        Real v = (Real)idy / (Real)height;

        Real aspect = (Real)width / (Real)height;

        Real3 ray = normalize(make_real3(2 * u - 1, (2 * v - 1) / aspect, 1.0));

        ray = transform3f(g_view, &ray);

        Real min = 0, max = FLT_MAX;

        if(intersectP(eye, ray, BBOX_MIN, BBOX_MAX, &min, &max))
        {
            r.screenCoord.x = idx;
            r.screenCoord.y = idy;

            r.dir_max.x = ray.x;
            r.dir_max.y = ray.y;
            r.dir_max.z = ray.z;

            r.origin_min.x = eye.x;
            r.origin_min.y = eye.y;
            r.origin_min.z = eye.z;

            r.setMin(min);
            r.setMax(max);

            r.rayWeight = 1;

            //uint id = r.screenCoord.y * width + r.screenCoord.x;

            TraceResult hitRes;
            traverse(g_tree, r.getOrigin(), r.getDir(), hitRes, *r.getMin(), *r.getMax(), cull_back);

            if(hitRes.isHit)
            {
                Material mat = g_geometry.getMaterial(hitRes.triIndex);
                //shade(hitRes, id, color, r, mat);

                Real3 hitPos = g_geometry.getTrianglelHitPos(hitRes.triIndex, hitRes.bary);
                Real3 normal = g_geometry.getTrianglelNormal(hitRes.triIndex, hitRes.bary);
// 
//                 bool isTrans = mat.isTransp();
//                bool isMirror = mat.isMirror();
// 
//                 Real3 dir = r.getDir();
                Real weight = r.rayWeight;
// 
                //if(isMirror)
                {
                    Real ratio = Reflectance(r.getDir(), normal, AIR_RI, mat.reflectionIndex(), mat.fresnel_r());
                    Real3 reflection = reflect(r.getDir(), normal);
                    r.rayWeight = ratio * weight * mat.reflectivity();
                    r.setDir(reflection);
                    //r.setOrigin(hitPos + RAY_HIT_NORMAL_DELTA * normal);
                    //r.clampToBBox();
                    //addRay(newReflectionRays, rayIndex, r);
                    //mask = 1;

                    float u = 0.5 + atan2(r.dir_max.z, r.dir_max.x) / (2 * PI);
                    float v = 0.5 - asin(r.dir_max.y) / PI;
                    float2 tc = make_float2(u, 1-v);
                    float4 c  = readTexture(0, tc);
                    c*=r.rayWeight;
                    c.x = 0.5 + .5 * dot(normal, make_real3(0,1,0)); c.y = 0; c.z = 0;
                    color[id].x = c.x;
                    color[id].y = c.y;
                    color[id].z = c.z;
                }
            }
            else
            {
                float u = 0.5 + atan2(r.dir_max.z, r.dir_max.x) / (2 * PI);
                float v = 1 - (0.5 - asin(r.dir_max.y) / PI);
                float2 tc = make_float2(u, v);
                float4 c  = readTexture(0, tc);
                color[id].x = c.x;
                color[id].y = c.y;
                color[id].z = c.z;
            }
        }
    }

//     __syncthreads();
// 
//     uint sum = __2DblockScan<blockSize>(shrdScanned, mask, linearBlockThreadId);
// 
//     __syncthreads();
// 
//     if(mask)
//     {
//         rayMemory[linearBlock * blockSize + sum - mask] = r;
//     }
// 
//      __syncthreads();
// 
//     if(linearBlockThreadId == blockSize-1)
//     {
//         const uint block = blockSize;
//        // __dpTraceRays<blockSize><<<1, blockSize>>>(rayMemory + linearBlock * blockSize, color, width, sum, 1);
//     }
}

__global__ void computeInitialRays(float4* color, Ray* rays, uint* rayMask, float3 eye, unsigned int width, unsigned int height, unsigned int N)
{
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;
    uint idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint id = idx + idy * blockDim.x * gridDim.x;

    if(idx >= width || idy >= height)
    {
        return;
    }
    
    Real u = (Real)idx / (Real)width;
    Real v = (Real)idy / (Real)height;

    Real aspect = (Real)width / (Real)height;

    Real3 ray = normalize(make_real3(2 * u - 1, (2 * v - 1) / aspect, 1.0));

    ray = transform3f(g_view, &ray);

    color[id] = make_float4(0,0,0,0); //clear buffer

    Real min = 0, max = FLT_MAX;
    
    if(!intersectP(eye, ray, BBOX_MIN, BBOX_MAX, &min, &max))
    {
        rayMask[id] = 0;
        return;
    } 

    rayMask[id] = 1;

    Ray r;

    r.screenCoord.x = idx;
    r.screenCoord.y = idy;
    
    r.dir_max.x = ray.x;
    r.dir_max.y = ray.y;
    r.dir_max.z = ray.z;

    r.origin_min.x = eye.x;
    r.origin_min.y = eye.y;
    r.origin_min.z = eye.z;

    r.setMin(min);
    r.setMax(max);

    r.rayWeight = 1;

    rays[id] = r;
}

__constant__ Real4 g_matrix[4]; 
__global__ void transform(Normal* n, Normal* newNormals, uint N)
{
    uint id = threadIdx.x + blockDim.x * blockIdx.x;

    if(id >= N)
    {
        return;
    }
    Normal _n = n[id];
    Real4 v = make_real4(_n.x, _n.y, _n.z, 0);
    newNormals[id] = transform4f(g_matrix, &v);
}

nutty::DeviceBuffer<Ray>* g_rays[2];
nutty::DeviceBuffer<uint>* g_rayMask;

nutty::DeviceBuffer<Ray>* g_refractionRays[2];
nutty::DeviceBuffer<uint>* g_refractionRayMask;

nutty::DeviceBuffer<Ray>* g_shadowRays[2];
nutty::DeviceBuffer<uint>* g_shadowRayMask;

nutty::DeviceBuffer<uint>* g_scannedRayMask;
nutty::DeviceBuffer<uint>* g_scannedSums;
nutty::DeviceBuffer<uint>* g_sums;

nutty::DeviceBuffer<Ray>* g_blockRayMemory;

nutty::DeviceBuffer<float4>* g_rawColors;
nutty::DeviceBuffer<float4>* g_blurColorsX;
nutty::DeviceBuffer<float4>* g_blurColorsY;
nutty::DeviceBuffer<float4>* g_downSampled;
nutty::DeviceBuffer<float>* g_luminance;
nutty::DeviceBuffer<float4>* g_hdColors;

RWTextureObject* g_blurredImage;

SurfaceObject* g_surface;
SurfaceObject* g_dsSurface;

int g_cpuCurrentLightCount;
rtLight* cpuLights[MAX_LIGHTS];

dim3 g_grid;
dim3 g_grp;
int g_recDepth = RECURSION;

uint g_width = 1;
uint g_height = 1;

extern "C" void RT_SetViewPort(unsigned int width, unsigned int height)
{
    g_width = width;

    g_height = height;

    g_grp.x = 32;
    g_grp.y = 32;
    g_grp.z = 1;
    
    g_grid.x = nutty::cuda::GetCudaGrid(width, g_grp.x);
    g_grid.y = nutty::cuda::GetCudaGrid(height, g_grp.y);
    g_grid.z = 1;
}

extern "C" void RT_TransformNormals(Normal* normals, Normal* newNormals, Real4* matrix, size_t start, uint N, hipStream_t stream)
{
    dim3 grid; dim3 group(256);

    grid.x = nutty::cuda::GetCudaGrid(N, group.x);

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbol(HIP_SYMBOL(g_matrix), matrix, 4 * sizeof(Real4), 0, hipMemcpyHostToDevice));
    transform<<<grid, group, 0, stream>>>(normals + start, newNormals + start, N);
}

const static int blurLine = 8;

extern "C" void RT_Init(unsigned int width, unsigned int height)
{
    g_shadowRays[0] = new nutty::DeviceBuffer<Ray>();
    g_shadowRays[1] = new nutty::DeviceBuffer<Ray>();

    g_rays[0] = new nutty::DeviceBuffer<Ray>();
    g_rays[1] = new nutty::DeviceBuffer<Ray>();

    g_refractionRays[0] = new nutty::DeviceBuffer<Ray>();
    g_refractionRays[1] = new nutty::DeviceBuffer<Ray>();

    g_rayMask = new nutty::DeviceBuffer<uint>();
    g_refractionRayMask = new nutty::DeviceBuffer<uint>();

    g_scannedRayMask = new nutty::DeviceBuffer<uint>();
    g_shadowRayMask = new nutty::DeviceBuffer<uint>();
    g_sums = new nutty::DeviceBuffer<uint>();
    g_scannedSums = new nutty::DeviceBuffer<uint>();

    g_blurredImage = new RWTextureObject();

    g_blockRayMemory = new nutty::DeviceBuffer<Ray>();

    int null = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(g_currentLightCount), &null, sizeof(int), 0, hipMemcpyHostToDevice);

    for(int i = 0; i < MAX_LIGHTS; ++i)
    {
        cpuLights[i] = new rtLight(i);
    }

    unsigned int maxRaysLastBranch = width * height * (1 << (MAX_RECURSION - 1));
    unsigned int maxRaysPerNode = width * height;

    g_rays[0]->Resize(maxRaysLastBranch);
    g_rays[1]->Resize(maxRaysLastBranch);
    g_rayMask->Resize(maxRaysLastBranch);
    g_scannedRayMask->Resize(maxRaysPerNode);

    g_refractionRays[0]->Resize(maxRaysPerNode);
    g_refractionRays[1]->Resize(maxRaysPerNode);
    g_refractionRayMask->Resize(maxRaysPerNode);

//     g_shadowRays[0]->Resize(maxRaysPerNode);
//     g_shadowRays[1]->Resize(maxRaysPerNode);
//     g_shadowRayMask->Resize(maxRaysPerNode);

    g_sums->Resize((2 * maxRaysPerNode) / 512);
    g_scannedSums->Resize((2 * maxRaysPerNode) / 512);

    RT_SetSunDir(make_float3(10,10,-10));

    int blurScale = blurLine * blurLine;

    g_rawColors = new nutty::DeviceBuffer<float4>(width * height);
    
    g_downSampled = new nutty::DeviceBuffer<float4>((width * height) / blurScale);
    g_blurColorsX = new nutty::DeviceBuffer<float4>((width * height) / blurScale);
    g_blurColorsY = new nutty::DeviceBuffer<float4>((width * height) / blurScale);

    g_hdColors = new nutty::DeviceBuffer<float4>(width * height);
    g_luminance = new nutty::DeviceBuffer<float>(width * height);

    float kernel[KERNEL_LENGTH] = {0.00000067, 0.00002292, 0.00019117, 0.00038771, 0.00019117, 0.00002292, 0.00000067};
    float sum = 0;
    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        sum += kernel[i];
    }

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        kernel[i] /= sum;
    }

    RT_EnvMapSale(1.0f);

    setConvolutionKernel(kernel);

    g_surface = new SurfaceObject();

    g_surface->Init(width, height);

    g_dsSurface = new SurfaceObject();

    g_dsSurface->Init(width / blurLine, height / blurLine);

    g_blurredImage->Init(width / blurLine, height / blurLine);
}

extern "C" void RT_Destroy(void)
{
    for(int i = 0; i < MAX_LIGHTS; ++i)
    {
        delete cpuLights[i];
    }

    delete g_blurredImage;
    delete g_surface;
    delete g_dsSurface;

    delete g_rays[0];
    delete g_rays[1];

    delete g_rawColors;
    delete g_blurColorsX;
    delete g_blurColorsY;
    delete g_downSampled;
    delete g_hdColors;
    delete g_luminance;

    delete g_refractionRays[0];
    delete g_refractionRays[1];

//     delete g_shadowRays[0];
//     delete g_shadowRays[1];

    delete g_rayMask;
    delete g_refractionRayMask;

    delete g_scannedRayMask;
    delete g_sums;
    delete g_scannedSums;
    delete g_shadowRayMask;
    delete g_blockRayMemory;
}

uint g_lastRays = 0;
std::stringstream g_info;

extern "C" void RT_GetRayInfo(std::string& info)
{
    info.clear();
    info += "\n\n";
    info += g_info.str();
}

extern "C" uint RT_GetLastRayCount(void)
{
    return g_lastRays;
}

__global__ void __spreadScannedSumsSingleT(uint* scanned, const uint* __restrict prefixSum, uint length)
{
    uint tileSize = 256;
    uint id = tileSize + blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= length)
    {
        return;
    }
    scanned[id] += prefixSum[id/tileSize];
}

uint compactRays(nutty::DeviceBuffer<uint>::iterator& maskBegin,
                 nutty::DeviceBuffer<Ray>::iterator& rayDstBegin, 
                 nutty::DeviceBuffer<Ray>::iterator& raySrcBegin, 
                 uint elementCount)
{
    hipMemsetAsync(g_scannedRayMask->GetPointer(), 0, sizeof(uint) * g_scannedRayMask->Size());

    //nutty::ExclusivePrefixSumScan(maskBegin, maskBegin + elementCount, g_scannedRayMask->Begin(), g_sums->Begin(), g_scannedSums->Begin());

    nutty::PrefixSumOp<CTuint> op;
    const uint BLOCK_SIZE = 256;
    const uint blockCount = 1;
    const uint elemsPerThread = 1;// * elemsPerBlock / BLOCK_SIZE;
    uint grid = nutty::cuda::GetCudaGrid(elementCount, BLOCK_SIZE);

    nutty::PrefixSumOp<uint> _op;
    __binaryGroupScan<BLOCK_SIZE><<<grid, BLOCK_SIZE>>>((const uint*)maskBegin(), (uint*)g_scannedRayMask->GetPointer(), g_sums->GetPointer(), _op, elementCount);
     __completeScan<1024><<<1, 1024>>>(g_sums->GetConstPointer(), g_scannedSums->GetPointer(), _op, grid);

    __spreadScannedSumsSingleT<<<grid-1, BLOCK_SIZE>>>(g_scannedRayMask->GetPointer(), g_scannedSums->GetPointer(), elementCount);

    nutty::Compact(rayDstBegin, raySrcBegin, raySrcBegin + elementCount, maskBegin, g_scannedRayMask->Begin(), 0U);

    auto it = g_scannedRayMask->Begin() + elementCount - 1;
    return *(g_scannedRayMask->Begin() + elementCount - 1) + *(maskBegin + elementCount - 1);
}

void traceRays(float4* colors, 
               int recDepth, 
               uint lastRayCount,
               uint width, uint height)
{
    //nutty::ZeroMem(*g_scannedRayMask);
    //nutty::ZeroMem(*g_sums);

    byte toggle = 0;
    std::queue<RayRange> q[2];

    RayRange ir;
    ir.begin = 0;
    ir.end = lastRayCount;
    q[0].push(ir);
    g_info.str("");
    for(int i = 0; i < recDepth; ++i)
    {
        uint src = toggle % 2;
        uint dst = (toggle+1) % 2;
        toggle ^= 1;

        //print("\nDepth=%d\n", i);
        g_info << "\nDepth=" << i << "\n";
        uint offset = (1 << (max(0, (recDepth - i - 2)))) * width * height;

        while(!q[src].empty())
        {
            RayRange range = q[src].front();
            q[src].pop();

            lastRayCount = compactRays(g_rayMask->Begin() + range.begin, g_rays[dst]->Begin() + range.begin, g_rays[src]->Begin() + range.begin, range.Length());
            
            g_lastRays += lastRayCount;
            //print("Range: from '%d' -> '%d' (L=%d) got '%d' Rays\n", range.begin, range.end, range.Length(), lastRayCount);
            g_info << "Range: '" << range.begin << "' ->'\n  "<< range.end << " got '" << lastRayCount << "' Rays\n";
            if(lastRayCount > 0)
            {
                uint blockSize = 256;
                dim3 g = nutty::cuda::GetCudaGrid(lastRayCount, blockSize);

                RayPair newShadowRays;
                newShadowRays.mask = g_shadowRayMask->Begin()();
                newShadowRays.rays = g_shadowRays[0]->Begin()();

                RayPair newRefractionRaysRays;
                newRefractionRaysRays.mask = g_refractionRayMask->Begin()();
                newRefractionRaysRays.rays = g_refractionRays[0]->Begin()();

                RayPair rayPairDst;
                rayPairDst.mask = g_rayMask->Begin()() + range.begin;
                rayPairDst.rays = g_rays[dst]->Begin()() + range.begin;

                nutty::ZeroMem<uint>(g_rayMask->Begin() + range.begin, g_rayMask->Begin() + range.end);
                nutty::ZeroMem(*g_refractionRayMask);
                nutty::ZeroMem(*g_shadowRayMask);

                if(i+1 == recDepth)
                {
                    _traceRays<0><<<g, blockSize>>>(colors, rayPairDst.rays, rayPairDst, newRefractionRaysRays, newShadowRays, width, height, lastRayCount);
                }
                else
                {
                    _traceRays<1><<<g, blockSize>>>(colors, rayPairDst.rays, rayPairDst, newRefractionRaysRays, newShadowRays, width, height, lastRayCount);

                    RayRange refRange;
                    refRange.begin = range.begin;
                    refRange.end = range.begin + lastRayCount;
                    q[dst].push(refRange);
                }

                //print("%d\n", hipDeviceSynchronize());
#if 0
                uint shadowRaysCount = compactRays(g_shadowRayMask->Begin(), g_shadowRays[1]->Begin(), g_shadowRays[0]->Begin(), lastRayCount);

                g_lastRays += shadowRaysCount;

                if(shadowRaysCount > 0)
                {
                    g = nutty::cuda::GetCudaGrid(shadowRaysCount, blockSize);
                    _traceShadowRays<<<g, blockSize>>>(colors, g_shadowRays[1]->Begin()(), width, height, shadowRaysCount);
                }
#endif

#if defined COMPUTE_REFRACTION
                if(i+1 < recDepth)
                {
                    uint refractionRaysCount = compactRays(g_refractionRayMask->Begin(), g_refractionRays[1]->Begin(), g_refractionRays[0]->Begin(), lastRayCount);
                    g_lastRays += refractionRaysCount;

                    if(refractionRaysCount > 0)
                    {
                        g = nutty::cuda::GetCudaGrid(refractionRaysCount, blockSize);
                        RayPair rayPair;
                        rayPair.mask = offset + g_rayMask->Begin()() + range.begin;
                        rayPair.rays = offset + g_rays[dst]->Begin()() + range.begin;
                        _traceRefractionRays<<<g, blockSize>>>(colors, g_refractionRays[1]->Begin()(), rayPair, width, height, refractionRaysCount);

                        RayRange refRange;
                        refRange.begin = offset + range.begin;
                        refRange.end = offset + range.begin + refractionRaysCount;
                        q[dst].push(refRange);
                    }
                }
#endif
            }
        }
    }
}

extern "C" void RT_Trace(float4* colors, const float3* view, float3 eye, BBox& bbox)
{
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbolAsync(HIP_SYMBOL(g_view), view, 3 * sizeof(float3), 0, hipMemcpyHostToDevice));

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbolAsync(HIP_SYMBOL(g_bbox), &bbox, sizeof(BBox), 0, hipMemcpyHostToDevice));

    //nutty::ZeroMem(*g_rayMask);

#ifndef MEASURING

    computeInitialRays<<<g_grid, g_grp>>>(g_rawColors->GetPointer(), g_rays[0]->Begin()(), g_rayMask->Begin()(), eye, g_width, g_height, g_width * g_height);

    g_lastRays = 0;

    traceRays(g_rawColors->GetPointer(), g_recDepth, g_width * g_height, g_width, g_height);

#else

    const uint block = 256;
    dim3 grp;
    dim3 grid;
    grp.x = 16;
    grp.y = 16;
    grp.z = 1;
    
    grid.x = nutty::cuda::GetCudaGrid(g_width, grp.x);
    grid.y = nutty::cuda::GetCudaGrid(g_height, grp.y);
    grid.z = 1;

    //g_blockRayMemory->Resize(block * (grid.y * grid.x));

    _dpTraceRays<block><<<grid, grp>>>(colors, g_blockRayMemory->GetPointer(), eye, g_width, g_height, g_width * g_height);

    DEVICE_SYNC_CHECK();
#endif


#ifndef MEASURING
//     getHDValues<<<g_grid, g_grp>>>(g_rawColors->GetPointer(), g_hdColors->GetPointer(), g_surface->surfObj, g_luminance->GetPointer(), g_width * g_height);
//     DEVICE_SYNC_CHECK();
// 
//     dim3 sdBlock;
//     sdBlock.x = 16;
//     sdBlock.y = 16;
// 
//     uint blurWidth = g_width / blurLine;
//     uint blurHeight = g_height / blurLine;
// 
//     dim3 sdGrid;
//     sdGrid.x = nutty::cuda::GetCudaGrid(blurWidth, sdBlock.x);
//     sdGrid.y = nutty::cuda::GetCudaGrid(blurHeight, sdBlock.y);
// 
//     scaleDown<blurLine><<<sdGrid, sdBlock>>>(g_downSampled->GetPointer(), g_surface->surfObj, g_width/blurLine, g_height/blurLine);
//     DEVICE_SYNC_CHECK();
// 
//     dim3 blocks(blurWidth / COLUMNS_BLOCKDIM_X, blurHeight / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
//     dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
// 
//     convolutionColumnsKernel<<<blocks, threads>>>(
//         g_blurColorsX->GetPointer(),
//         g_downSampled->GetPointer(),
//         blurWidth,
//         blurHeight,
//         blurWidth
//     );
// 
//     dim3 rowblocks(blurWidth / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), blurHeight / ROWS_BLOCKDIM_Y);
//     dim3 rowthreads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
// 
//     convolutionRowsKernel<<<rowblocks, rowthreads>>>(
//         g_blurColorsY->GetPointer(),
//         g_blurColorsX->GetPointer(),
//         blurWidth,
//         blurHeight,
//         blurWidth
//     );
// 
//     cudaMemcpyToArrayAsync(g_blurredImage->cuArray, 0, 0, g_blurColorsY->GetPointer(), g_blurColorsY->Size() * sizeof(float4), hipMemcpyDeviceToDevice);

    //nutty::Reduce(g_luminance->Begin(), g_luminance->End(), nutty::binary::Plus<float>(), 0.0f);

    addHDValues<<<g_grid, g_grp>>>(colors, g_blurredImage->tex, g_rawColors->GetPointer() , g_width * g_height, blurLine, g_width, g_height, g_luminance->GetPointer());
#endif
}

extern "C" float4* RT_GetRawColors    (
    CTuint* width,
    CTuint* height
    )
{
    *width = g_width;
    *height = g_height;
    return g_rawColors->GetPointer();
}
 
extern "C" void RT_BindTree(TreeNodes& tree)
{
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbol(HIP_SYMBOL(g_tree), &tree, sizeof(TreeNodes), 0, hipMemcpyHostToDevice));
}

extern "C" void RT_SetSunDir(const float3& pos)
{
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbol(HIP_SYMBOL(g_sunPos), &pos, sizeof(float3), 0, hipMemcpyHostToDevice));
}

extern "C" void RT_BindGeometry(Triangles& tries)
{
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbol(HIP_SYMBOL(g_geometry), &tries, sizeof(Triangles), 0, hipMemcpyHostToDevice));
}

extern "C" int RT_GetRecDepth(void)
{
    return g_recDepth;
}

extern "C" void RT_SetRecDepth(int d)
{
    g_recDepth =  d < 1 ? 1 : (d > MAX_RECURSION ? MAX_RECURSION : d);
}

extern "C" void RT_IncDepth(void)
{
    RT_SetRecDepth(g_recDepth+1);
}

extern "C" void RT_DecDepth(void)
{
    RT_SetRecDepth(g_recDepth-1);
}

extern "C" void RT_SetShader(int shaderId)
{
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbol(HIP_SYMBOL(g_shader), &shaderId, sizeof(int), 0, hipMemcpyHostToDevice));
}

extern "C" void RT_AddLight(RT_Light_t* light)
{
    if(g_cpuCurrentLightCount >= MAX_LIGHTS)
    {
        *light = NULL;
        return;
    }

    *light = cpuLights[g_cpuCurrentLightCount];

    g_cpuCurrentLightCount++;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(g_currentLightCount), &g_cpuCurrentLightCount, sizeof(int), 0, hipMemcpyHostToDevice);
}

extern "C" void RT_EnvMapSale(float scale)
{
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpyToSymbol(HIP_SYMBOL(g_envMapScale), &scale, sizeof(float), 0, hipMemcpyHostToDevice));
}

extern "C" void RT_BindTextureAtlas(const hipArray_t array)
{
//     hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint2>();
//     g_atlas.normalized = 1;
//     g_atlas.filterMode = hipFilterModePoint;
//     g_atlas.addressMode[0] = hipAddressModeClamp;
//     g_atlas.addressMode[1] = hipAddressModeClamp;
//     g_atlas.addressMode[2] = hipAddressModeClamp;
// 
//     CUDA_RT_SAFE_CALLING_NO_SYNC(hipBindTextureToArray(g_atlas, array, channelDesc));
}

void __device__ Ray::clampToBBox(void)
{
    setMin(0);
    setMax(1000.0);
    Real3 o = {origin_min.x, origin_min.y, origin_min.z};
    Real3 d = {dir_max.x, dir_max.y, dir_max.z};

    intersectP(o, d, BBOX_MIN, BBOX_MAX, getMin(), getMax());
}
